#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>   
#include <cuPrintf.cu>
#include <sm_12_atomic_functions.h>


//#define VERTEX_INSERTION_QUALITY_THRESHOLD 0.26
#define VERTEX_INSERTION_QUALITY_THRESHOLD 0.65
//#define VERTEX_INSERTION_QUALITY_THRESHOLD 0.45 
#define MINIMPROVEMENT 1.0e-6

#define MAXCAVITYFACES 30
#define MAXCAVITYTETS 50
#define MAXCAVITYEDGES 150
#define MAXSTACKTET 60
#define MAXSTACKFACE 90

#define HUGEFLOAT  1.0e10
#define HUGEQUAL 1.0e10
#define MINFACING 1.0e-7
#define MINFLIPIMPROVE 1.0e-6
#define CAVDEPTHLIMIT 6
/*vertex insert*/
#define EDGELABEL 0
//#define TETLABEL 1
#define CAVLABEL 0
#define ANTICAVLABEL 1
#define NOLABEL 2
#define DEPTHTABLESIZE 10
#define NOCAVITYTET -1
#define NOCAVITYFACE -1
#define GHOSTTET -1
#define imin(a,b) (a<b?a:b)

const int BlockPerGrid = 4096;
const int ThreadPerBlock = 128;
//const int ThreadsPerBlock = 256;

/************************************************************************/
/*                     cuPrint initialization                           */
/************************************************************************/

bool InitGPUSet()  
{  
	char GPU[100] = "GPU: ";  
	hipDeviceProp_t tCard;  
	int num = 0;  
	if(hipSuccess == hipGetDeviceCount(&num))  
	{  
		for(int i = 0; i < num; ++ i)  
		{  
			hipSetDevice(i);  
			hipGetDeviceProperties(&tCard, i);  
			puts(strcat(GPU , tCard.name));//���صľ������Ӻ�Ľ��,ҲΪ���Ƕ��ʹ���ṩ������   
		}  
	}  
	else  return false;  
	return true;  
}  
bool cuPrintInit()  
{  
	hipError_t err = cudaPrintfInit();  
	if(0 != strcmp("no error", hipGetErrorString(err)))  return false;  
	return true;  
}  
__global__ void displayGPU_demo()  
{  
	int bsize = blockDim.x;  
	int bid = blockIdx.x;  
	int tid = bid * bsize + threadIdx.x;  
	cuPrintf("��ǰִ��kernel�� block ���:\t%d\n", bid);  
	cuPrintf("��ǰִ��kernel�� thread �ڵ�ǰ���б��:\t%d\n", threadIdx.x);  
	cuPrintf("��ǰִ��kernel�� thread ȫ�ֱ��:\t%d\n", tid);  
	cuPrintf("thread over\n\n");  
}

// ʾ����������������һ��else�Ｔ��
extern "C" void testCudaPrintf()
{
	if(!InitGPUSet())  puts("device is not ready!");  
	else if(!cuPrintInit())  puts("device is not ready!");  
	else  
	{  
		displayGPU_demo<<<2, 3>>>();  
		cudaPrintfDisplay(stdout, true);//true�������һ��block�ĵڼ���thread��ִ�б��������䣬���磺[blockID, threadID]��false�����   
		cudaPrintfEnd();  
	} 
}

/*************************end of cuPrintf initialization ****************************/

/************************************************************************/
/* Data structure                                                       */
/************************************************************************/
struct cu_point
{
	float vec3f[3];
};

struct cu_tet
{
	int v[4];
};


struct cu_halfface
{
	int pointhandle[3];
	int face;
};

struct cu_halfedge
{
	int edgehandle;
	int fromv;
	int tov;
};

struct cu_face
{
	int hf[2];
};

struct cu_flip23face
{
	int hf[2];
	float quality;  // worst quality of incident tets
	float val;      // the improvement of flip23
};



struct CavityFace
{
	int handle;
	float quality;
	int child;
	bool inH;
};

struct CavityEdge
{
	float qual;
	int label;
	int parent;
	int child;
	int childnum;
};

struct CavityTet
{
	int handle;
	float quality;
	int depth;
	CavityFace outfaces[4];
	int outfacesize;
	int parents[3];
	int label;
};

struct cu_InsertTet
{
	int v[4];
	int deletetet[MAXCAVITYTETS];
	int deletetetcnt;
	int cavityface[MAXCAVITYFACES];
	int cavityfacecnt;
	float insertpoint[3];
	float quality;
	float val;
};

struct cu_edge
{
	bool is_boundary;
	int halfedgecnt;
	int halfedge[100];
	float quality;
	float val;
};


struct cu_flip32edge
{
	int p[2];
	int tet[3];
	float quality;  // worst quality of tets
	float val;      // the improvement of flip32
	int order;
};

struct cu_tetra
{
	bool isboundary;
	int v[4];
	float quality;
	float val;
	int fliptype;
	int strategy;
	int newflipvertex;
	int newflipface[4];
	int tet[4];
	int flippoint[2];
};

template<class T>
struct queue
{
	queue()
	{
		initialize();
	}

	~queue()
	{
		node *tmp;
		while(tmp!=NULL)
		{
			tmp=head;
			head=head->next;
			delete tmp;
			tmp=NULL;
		}
		delete cur;
	}

	void initialize()
	{
		head=new node(-1);
		cur=head;
		len=0;
	}

	bool empty() const
	{
		return len==0;
	}

	T& back()
	{
		return cur->val;
	}

	const T& back() const
	{
		return back();
	}

	void pop()
	{
		if(head->next==cur)
		{
			delete head->next;
			head->next=NULL;
		}else
		{
			node* tmp=head->next;
			head->next=tmp->next;
			delete tmp;
		}
		--len;
	}

	T& front()
	{
		return head->next->val;
	}

	const T& front() const
	{
		return front();
	}

	void push(const T& val)
	{
		node *tmp=new node(val);
		cur->next=tmp;
		cur=tmp;
		++len;
	}

	int size()
	{
		return len;
	}

	typedef struct node1
	{
		node1 *next;
		T val;
		node1(T v):val(v),next(NULL){}
	}node;

	int len;
	node *head;
	node *cur;

};
/********************************* end of data structure**************************************/

/****************************** Function Statement *********************************/
extern "C" void cuda_tetquality(float *points, int pointcnt, int *meshtets, int tetcnt, int qualmeasure, float &minqual);

/****************************End of Function Statement******************************/

/************************************************************************/
/*  tetrahedron quality calculate                                       */
/************************************************************************/
/* types of quality measures that may be used */
extern enum CudaTetQualityMetrics
{
	CUDA_QUAL_MINSINE,
	CUDA_QUAL_BIASEDMINSINE,
	CUDA_QUAL_RADIUSRATIO,
	CUDA_QUAL_VLRMS3RATIO,
	CUDA_QUAL_MEANSINE,
	CUDA_QUAL_MINSINEANDEDGERATIO,
	CUDA_QUAL_WARPEDMINSINE,
	CUDA_QUAL_MINANGLE,
	CUDA_QUAL_MAXANGLE
};

__device__ void vector_cross(float *a, float *b, float *c)
{
	c[0] = a[1] * b[2] - a[2] * b[1];
	c[1] = a[2] * b[0] - a[0] * b[2];
	c[2] = a[0] * b[1] - a[1] * b[0];
}

__device__ float vector_dot(float *a, float *b)
{
	return a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
}

__device__ void vector_add(float *a, float *b, float *c)
{
	c[0] = a[0] + b[0];
	c[1] = a[1] + b[1];
	c[2] = a[2] + b[2];
}

__device__ void vector_minus(float *a, float *b, float *c)
{
	c[0] = a[0] - b[0];
	c[1] = a[1] - b[1];
	c[2] = a[2] - b[2];
}

__device__ float minsine(float point[4][3])
{
	float t[3], u[3], v[3]; /* tet vectors */
	float temp[3];
    float edgelength[3][4]; /* the lengths of each of the edges of the tet */
    float facenormal[4][3]; /* the normals of each face of the tet */
    float dx, dy, dz;       /* intermediate values of edge lengths */
    float facearea2[4];     /* areas of the faces of the tet */
    float pyrvolume;        /* volume of tetrahedron */
    float sine2, minsine2;  /* the sine (squared) of the dihedral angle */
    int i, j, k, l;          /* loop indices */
    
    /* calculate the volume*6 of the tetrahedron */
	vector_minus(point[1], point[0], t);
	vector_minus(point[2], point[0], u);
	vector_minus(point[3], point[0], v);
	vector_cross(t, u, temp);
    pyrvolume = vector_dot(temp, v);
    
    /* if the volume is zero, the quality is zero, no reason to continue */
    if (pyrvolume == 0.0)
        return 0.0;
    
    /* for each vertex/face of the tetrahedron */
    for (i = 0; i < 4; i++) {
        j = (i + 1) & 3;
        if ((i & 1) == 0) {
            k = (i + 3) & 3;
            l = (i + 2) & 3;
        } else {
            k = (i + 2) & 3;
            l = (i + 3) & 3;
        }
        
        /* compute the normal for each face */
        facenormal[i][0] =
            (point[k][1] - point[j][1]) * (point[l][2] - point[j][2]) -
            (point[k][2] - point[j][2]) * (point[l][1] - point[j][1]);
        facenormal[i][1] =
            (point[k][2] - point[j][2]) * (point[l][0] - point[j][0]) -
            (point[k][0] - point[j][0]) * (point[l][2] - point[j][2]);
        facenormal[i][2] =
            (point[k][0] - point[j][0]) * (point[l][1] - point[j][1]) -
            (point[k][1] - point[j][1]) * (point[l][0] - point[j][0]);
            
        /* compute (2 *area)^2 for this face */
        facearea2[i] = facenormal[i][0] * facenormal[i][0] +
            facenormal[i][1] * facenormal[i][1] +
            facenormal[i][2] * facenormal[i][2];
        
        /* compute edge lengths (squared) */
        for (j = i + 1; j < 4; j++) {
            dx = point[i][0] - point[j][0];
            dy = point[i][1] - point[j][1];
            dz = point[i][2] - point[j][2];
            edgelength[i][j] = dx * dx + dy * dy + dz * dz;
        }
    }
    
    minsine2 = HUGEQUAL;     /* start with absurdly big value for sine */
    
    /* for each edge in the tetrahedron */
    for (i = 0; i < 3; i++) {
        for (j = i + 1; j < 4; j++) {
            k = (i > 0) ? 0 : (j > 1) ? 1 : 2;
            l = 6 - i - j - k;
            
            /* compute the expression for minimum sine, squared, over 4 
               The reason it's over 4 is because the area values we have
               are actually twice the area squared */
            /* if either face area is zero, the sine is zero */
            if (facearea2[k] > 0 && facearea2[l] > 0)
            {
                sine2 = edgelength[i][j] / (facearea2[k] * facearea2[l]);
            }
            else
            {
                sine2 = 0.0;
            }
            
            /* update minimum sine */
            if (sine2 < minsine2)
            {
                minsine2 = sine2;
            }
        }
    }
    
    return sqrt(minsine2) * pyrvolume;
}


__device__ float minsine(float p1[3], float p2[3], float p3[3], float p4[3])
{
	float t[3], u[3], v[3]; /* tet vectors */
	float temp[3];
    float edgelength[3][4]; /* the lengths of each of the edges of the tet */
    float facenormal[4][3]; /* the normals of each face of the tet */
    float dx, dy, dz;       /* intermediate values of edge lengths */
    float facearea2[4];     /* areas of the faces of the tet */
    float pyrvolume;        /* volume of tetrahedron */
    float sine2, minsine2;  /* the sine (squared) of the dihedral angle */
    int i, j, k, l;          /* loop indices */
	float point[4][3];

	for (i = 0; i < 3; i++)
	{
		point[0][i] = p1[i];
		point[1][i] = p2[i];
		point[2][i] = p3[i];
		point[3][i] = p4[i];
	}
    
    /* calculate the volume*6 of the tetrahedron */
	vector_minus(point[1], point[0], t);
	vector_minus(point[2], point[0], u);
	vector_minus(point[3], point[0], v);
	vector_cross(t, u, temp);
    pyrvolume = vector_dot(temp, v);
    
    /* if the volume is zero, the quality is zero, no reason to continue */
    if (pyrvolume == 0.0)
        return 0.0;
    
    /* for each vertex/face of the tetrahedron */
    for (i = 0; i < 4; i++) {
        j = (i + 1) & 3;
        if ((i & 1) == 0) {
            k = (i + 3) & 3;
            l = (i + 2) & 3;
        } else {
            k = (i + 2) & 3;
            l = (i + 3) & 3;
        }
        
        /* compute the normal for each face */
        facenormal[i][0] =
            (point[k][1] - point[j][1]) * (point[l][2] - point[j][2]) -
            (point[k][2] - point[j][2]) * (point[l][1] - point[j][1]);
        facenormal[i][1] =
            (point[k][2] - point[j][2]) * (point[l][0] - point[j][0]) -
            (point[k][0] - point[j][0]) * (point[l][2] - point[j][2]);
        facenormal[i][2] =
            (point[k][0] - point[j][0]) * (point[l][1] - point[j][1]) -
            (point[k][1] - point[j][1]) * (point[l][0] - point[j][0]);
            
        /* compute (2 *area)^2 for this face */
        facearea2[i] = facenormal[i][0] * facenormal[i][0] +
            facenormal[i][1] * facenormal[i][1] +
            facenormal[i][2] * facenormal[i][2];
        
        /* compute edge lengths (squared) */
        for (j = i + 1; j < 4; j++) {
            dx = point[i][0] - point[j][0];
            dy = point[i][1] - point[j][1];
            dz = point[i][2] - point[j][2];
            edgelength[i][j] = dx * dx + dy * dy + dz * dz;
        }
    }
    
    minsine2 = HUGEQUAL;     /* start with absurdly big value for sine */
    
    /* for each edge in the tetrahedron */
    for (i = 0; i < 3; i++) {
        for (j = i + 1; j < 4; j++) {
            k = (i > 0) ? 0 : (j > 1) ? 1 : 2;
            l = 6 - i - j - k;
            
            /* compute the expression for minimum sine, squared, over 4 
               The reason it's over 4 is because the area values we have
               are actually twice the area squared */
            /* if either face area is zero, the sine is zero */
            if (facearea2[k] > 0 && facearea2[l] > 0)
            {
                sine2 = edgelength[i][j] / (facearea2[k] * facearea2[l]);
            }
            else
            {
                sine2 = 0.0;
            }
            
            /* update minimum sine */
            if (sine2 < minsine2)
            {
                minsine2 = sine2;
            }
        }
    }
    
    return sqrt(minsine2) * pyrvolume;
}


__device__ float tetquality(float *points, int *meshtets, int tetcnt, int tetidx, int qualmeasure)
{
	if (tetidx < 0 || tetidx > tetcnt-1)
		return -1.0;

	float point[4][3];
	float quality = 0.0; /* the quality of this tetrahedron */
	int pidx;

	for (int j = 0; j < 4; j++)
	{
		pidx = meshtets[4*tetidx+j];
		point[j][0] = points[3*pidx];
		point[j][1] = points[3*pidx+1];
		point[j][2] = points[3*pidx+2];
	}

	switch (qualmeasure)
	{
	case CUDA_QUAL_MINSINE:
		quality = minsine(point);
		break;
	case CUDA_QUAL_BIASEDMINSINE:
		//quality = biasedminsine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MEANSINE:
		//quality = meansine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MINSINEANDEDGERATIO:
		//quality = minsineandedgeratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_RADIUSRATIO:
		//quality = radiusratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_VLRMS3RATIO:
		//quality = vlrms3ratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_WARPEDMINSINE:
		//quality = warpedminsine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MINANGLE:
		//quality = minmaxangle(p1, p2, p3, p4, false);
		break;
	case CUDA_QUAL_MAXANGLE:
		//quality = minmaxangle(p1, p2, p3, p4, true);
		break;
	}
	return quality;
}

__device__ float tetquality(float p1[3], float p2[3], float p3[3], float p4[3], int qualmeasure)
{
	float quality = 0.0; /* the quality of this tetrahedron */
	switch (qualmeasure)
	{
	case CUDA_QUAL_MINSINE:
		quality = minsine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_BIASEDMINSINE:
		//quality = biasedminsine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MEANSINE:
		//quality = meansine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MINSINEANDEDGERATIO:
		//quality = minsineandedgeratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_RADIUSRATIO:
		//quality = radiusratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_VLRMS3RATIO:
		//quality = vlrms3ratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_WARPEDMINSINE:
		//quality = warpedminsine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MINANGLE:
		//quality = minmaxangle(p1, p2, p3, p4, false);
		break;
	case CUDA_QUAL_MAXANGLE:
		//quality = minmaxangle(p1, p2, p3, p4, true);
		break;
	}
	return quality;
}


__device__ float tetquality(float point[4][3], int qualmeasure)
{
	float quality = 0.0; /* the quality of this tetrahedron */
	switch (qualmeasure)
	{
	case CUDA_QUAL_MINSINE:
		quality = minsine(point);
		break;
	case CUDA_QUAL_BIASEDMINSINE:
		//quality = biasedminsine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MEANSINE:
		//quality = meansine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MINSINEANDEDGERATIO:
		//quality = minsineandedgeratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_RADIUSRATIO:
		//quality = radiusratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_VLRMS3RATIO:
		//quality = vlrms3ratio(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_WARPEDMINSINE:
		//quality = warpedminsine(p1, p2, p3, p4);
		break;
	case CUDA_QUAL_MINANGLE:
		//quality = minmaxangle(p1, p2, p3, p4, false);
		break;
	case CUDA_QUAL_MAXANGLE:
		//quality = minmaxangle(p1, p2, p3, p4, true);
		break;
	}
	return quality;
}

__device__ float mintetquality(float *points, int *meshtets, int tetcnt, int qualmeasure)
{
	float qual, minqual;
	float tetpoint[4][3];
	int pidx;
	minqual = HUGEQUAL;
	for (int i = 0; i < tetcnt; i++)
	{
		// get tetra points
		for (int j = 0; j < 4; j++)
		{
			pidx = meshtets[4*i+j];
			tetpoint[j][0] = points[3*pidx];
			tetpoint[j][1] = points[3*pidx+1];
			tetpoint[j][2] = points[3*pidx+2];
		}
		// calculate tetra quality
		qual = tetquality(tetpoint, qualmeasure);
		// fetch the minimum quality value
		if (minqual > qual)
			minqual = qual;
	}
	return minqual;
}

__device__ float minstackquality(float *points, int *meshtets, int *tethandlestack, int stackcnt, int qualmeasure)
{
	float qual, minqual;
	float tetpoint[4][3];
	int tetidx;
	int pidx;

	minqual = HUGEQUAL;
	for (int i = 0; i < stackcnt; i++)
	{
		tetidx = tethandlestack[i];
		for (int j = 0; j < 4; j ++)
		{
			pidx = meshtets[4*tetidx+j];
			tetpoint[j][0] = points[3*pidx];
			tetpoint[j][1] = points[3*pidx+1];
			tetpoint[j][2] = points[3*pidx+2];
		}
		qual = tetquality(tetpoint, qualmeasure);
		if (minqual > qual)
			minqual = qual;
	}
	return minqual;
}

__device__ float minstackquality(float *tetrapoints, int tetracnt, int qualmeasure)
{
	float qual, minqual;
	float tetpoint[4][3];

	minqual = HUGEQUAL;
	for (int i = 0; i < tetracnt; i++)
	{
		for (int j = 0; j < 4; j ++)
		{
			tetpoint[j][0] = tetrapoints[3*(4*i+j)];
			tetpoint[j][1] = tetrapoints[3*(4*i+j)+1];
			tetpoint[j][2] = tetrapoints[3*(4*i+j)+2];
		}
		qual = tetquality(tetpoint, qualmeasure);
		if (minqual > qual)
			minqual = qual;
	}
	return minqual;
}

__global__ void cuda_mintetquality(float *points, int *meshtets, int tetcnt, int qualmeasure, float *tetqual)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	float tetpoint[4][3];
	float qual;
	int pidx;

	while(tid < tetcnt)
	{
		// if the tet is invalid
		if (meshtets[4*tid] == -1)
		{
			tetqual[tid] = 1.0;
			tid += offset;
			continue;
		}

		// get tetra points
		for (int j = 0; j < 4; j++)
		{
			pidx = meshtets[4*tid+j];
			tetpoint[j][0] = points[3*pidx];
			tetpoint[j][1] = points[3*pidx+1];
			tetpoint[j][2] = points[3*pidx+2];
		}
		// calculate tetra quality
		qual = tetquality(tetpoint, qualmeasure);
		tetqual[tid] = qual;

		tid += offset;
	}
}

void cutetramesh_quality(float *points, int pointcnt, cu_tetra *ctetra, int tetcnt, int qualmeasure, float &minqual)
{
	int *meshtets;
	meshtets = new int[4*tetcnt];
	for (int i = 0; i < tetcnt; i++)
	{
		meshtets[4*i]   = ctetra[i].v[0];
		meshtets[4*i+1] = ctetra[i].v[1];
		meshtets[4*i+2] = ctetra[i].v[2];
		meshtets[4*i+3] = ctetra[i].v[3];
	}

	float *dev_points;
	int *dev_meshtets;
	float *dev_tetqual;

	hipMalloc((void**)&dev_points, 3*pointcnt*sizeof(float));
	hipMalloc((void**)&dev_meshtets, 4*tetcnt*sizeof(int));
	hipMalloc((void**)&dev_tetqual, tetcnt*sizeof(float));
	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_meshtets, meshtets, 4*tetcnt*sizeof(int), hipMemcpyHostToDevice);

	int blocks = imin(tetcnt, BlockPerGrid);
	cuda_mintetquality<<<blocks,(tetcnt+blocks-1)/blocks>>>(dev_points, dev_meshtets, tetcnt, qualmeasure, dev_tetqual);

	float *tetqual;
	tetqual = new float[tetcnt];
	hipMemcpy(tetqual, dev_tetqual, tetcnt*sizeof(float), hipMemcpyDeviceToHost);

	minqual = HUGEQUAL;
	for (int i = 0; i < tetcnt; i++)
		if (tetqual[i] < minqual)
			minqual = tetqual[i];

	hipFree(dev_points);
	hipFree(dev_meshtets);
	hipFree(dev_tetqual);
	delete [] tetqual;
	delete [] meshtets;
}

extern "C" void cuda_tetquality(float *points, int pointcnt, int *meshtets, int tetcnt, int qualmeasure, float &minqual)
{
	float *dev_points;
	int *dev_meshtets;
	float *dev_tetqual;

	hipMalloc((void**)&dev_points, 3*pointcnt*sizeof(float));
	hipMalloc((void**)&dev_meshtets, 4*tetcnt*sizeof(int));
	hipMalloc((void**)&dev_tetqual, tetcnt*sizeof(float));
	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_meshtets, meshtets, 4*tetcnt*sizeof(int), hipMemcpyHostToDevice);

	int blocks = imin(tetcnt, BlockPerGrid);
	cuda_mintetquality<<<blocks,(tetcnt+blocks-1)/blocks>>>(dev_points, dev_meshtets, tetcnt, qualmeasure, dev_tetqual);

	float *tetqual;
	tetqual = new float[tetcnt];
	hipMemcpy(tetqual, dev_tetqual, tetcnt*sizeof(float), hipMemcpyDeviceToHost);

	minqual = HUGEQUAL;
	for (int i = 0; i < tetcnt; i++)
		if (tetqual[i] > MINFLIPIMPROVE  && tetqual[i] < minqual)
			minqual = tetqual[i];

	hipFree(dev_points);
	hipFree(dev_meshtets);
	hipFree(dev_tetqual);
	delete [] tetqual;
}
/********************* end of tetrahedron quality ************************/

/************************ Parallel vertex smoothing ***************************/
__global__ void laplacianSmoothing(float *points, int pointcnt, int *neighbour, int *neighbourcnt, int largestn,
								   int *incidenttet, int *incidenttetcnt, int largesttet,
								   int *meshtets, int tetcnt)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	float point[3];
	float oldpoint[3];
	float qualbefore, qualafter;

	while(tid < pointcnt)
	{
		// calculate original quality
		qualbefore = minstackquality(points, meshtets, &incidenttet[largesttet*tid], incidenttetcnt[tid], CUDA_QUAL_MINSINE);

		// fetch neighbour count
		int ncnt = neighbourcnt[tid];
		int nbr;
		point[0] = point[1] = point[2] = 0.0;

		// save original point coordinates
		oldpoint[0] = points[3*tid];
		oldpoint[1] = points[3*tid+1];
		oldpoint[2] = points[3*tid+2];

		// calculate new point coordinates
		for (int i = 0; i < ncnt; i++)
		{
			nbr = neighbour[tid*largestn+i];
			point[0] += points[3*nbr];
			point[1] += points[3*nbr+1];
			point[2] += points[3*nbr+2];
		}
		point[0] /= ncnt;
		point[1] /= ncnt;
		point[2] /= ncnt;

		// set new point
		points[3*tid]   = point[0];
		points[3*tid+1] = point[1];
		points[3*tid+2] = point[2];

		// calculate new tetra quality
		qualafter = minstackquality(points, meshtets, &incidenttet[largesttet*tid], incidenttetcnt[tid], CUDA_QUAL_MINSINE);
		if (qualafter < qualbefore)
		{
			points[3*tid]   = oldpoint[0];
			points[3*tid+1] = oldpoint[1];
			points[3*tid+2] = oldpoint[2];
		}

		tid += offset;
	}
}


// grouped smoothing��only smooth one group at one time
__global__ void group_laplacianSmoothing(float *points, int pointcnt, int *neighbour, int *neighbourcnt, int largestn,
										  int *pointgroup, int activegroup,int *incidenttet, int *incidenttetcnt, 
										  int largesttet, int *meshtets, int tetcnt)
{	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	float point[3];
	float oldpoint[3];
	float qualbefore, qualafter;

	while(tid < pointcnt && pointgroup[tid] == activegroup)
	{
		// calculate original quality
		qualbefore = minstackquality(points, meshtets, &incidenttet[largesttet*tid], incidenttetcnt[tid], CUDA_QUAL_MINSINE);

		// fetch neighbour count
		int ncnt = neighbourcnt[tid];
		int nbr;
		point[0] = point[1] = point[2] = 0.0;

		// save original point coordinates
		oldpoint[0] = points[3*tid];
		oldpoint[1] = points[3*tid+1];
		oldpoint[2] = points[3*tid+2];

		// calculate new point coordinates
		for (int i = 0; i < ncnt; i++)
		{
			nbr = neighbour[tid*largestn+i];
			point[0] += points[3*nbr];
			point[1] += points[3*nbr+1];
			point[2] += points[3*nbr+2];
		}
		point[0] /= ncnt;
		point[1] /= ncnt;
		point[2] /= ncnt;

		// set new point
		points[3*tid]   = point[0];
		points[3*tid+1] = point[1];
		points[3*tid+2] = point[2];

		// calculate new tetra quality
		qualafter = minstackquality(points, meshtets, &incidenttet[largesttet*tid], incidenttetcnt[tid], CUDA_QUAL_MINSINE);
		if (qualafter < qualbefore)
		{
			points[3*tid]   = oldpoint[0];
			points[3*tid+1] = oldpoint[1];
			points[3*tid+2] = oldpoint[2];
		}

		tid += offset;
	}
}

extern "C" void cuda_vertexSmoothing(float *points, int pointcnt, int *hneighbour, int *neighbourcnt, int *pointgroup, int pointgroupcnt, 
									 float *newpoints, int *hincidenttet, int *incidenttetcnt, int *meshtets, int tetcnt, 
									 int largestn, int largesttet, int smoothpasscnt, float &time)
{
	int hncnt, hntet;
	hncnt = pointcnt * largestn;
	hntet = pointcnt * largesttet;

	// �����豸�洢�ռ�
	int *dev_neighbour;
	int *dev_neighbourcnt;
	float *dev_points;
	int *dev_incidenttet;
	int *dev_incidenttetcnt;
	int *dev_meshtets;
	int *dev_pointgroup;
	bool *dev_pointsmoothed;

	hipMalloc((void**)&dev_points, 3*pointcnt*sizeof(float));
	hipMalloc((void**)&dev_neighbour, hncnt*sizeof(int));
	hipMalloc((void**)&dev_neighbourcnt, pointcnt*sizeof(int));
	hipMalloc((void**)&dev_incidenttet, hntet*sizeof(int));
	hipMalloc((void**)&dev_incidenttetcnt, pointcnt*sizeof(int));
	hipMalloc((void**)&dev_meshtets, 4*tetcnt*sizeof(int));
	hipMalloc((void**)&dev_pointgroup, pointcnt*sizeof(int));
	hipMalloc((void**)&dev_pointsmoothed, pointcnt*sizeof(bool));

	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_neighbour, hneighbour, pointcnt*largestn*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_neighbourcnt, neighbourcnt, pointcnt*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_incidenttet, hincidenttet, hntet*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_incidenttetcnt, incidenttetcnt, pointcnt*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_meshtets, meshtets, 4*tetcnt*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_pointgroup, pointgroup, pointcnt*sizeof(int), hipMemcpyHostToDevice);
	hipMemset(dev_pointsmoothed, 0, pointcnt*sizeof(bool));

	int blocks = imin(pointcnt, BlockPerGrid);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// ���ú˺���
	//int blockPerGrid = (threadsPerBlock+pointcnt)/threadsPerBlock;
	while (smoothpasscnt)
	{
		for (int passidx = 0; passidx < smoothpasscnt; passidx ++)
		{
			//�� groupcnt �η��� kernel
			for (int i = 0; i < pointgroupcnt; i++)
			{
				group_laplacianSmoothing<<<blocks, (pointcnt+blocks-1)/blocks>>>(dev_points, pointcnt, dev_neighbour, dev_neighbourcnt, largestn, dev_pointgroup, i, 
					dev_incidenttet, dev_incidenttetcnt, largesttet, dev_meshtets, tetcnt);
			}
		}
		--smoothpasscnt;
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elaspsedTime;
	hipEventElapsedTime(&elaspsedTime, start, stop);
	time = elaspsedTime;

	hipMemcpy(newpoints, dev_points, 3*pointcnt*sizeof(float), hipMemcpyDeviceToHost);

	bool *pointsmoothed;
	pointsmoothed = new bool[pointcnt];
	hipMemcpy(pointsmoothed, dev_pointsmoothed, pointcnt*sizeof(bool), hipMemcpyDeviceToHost);


	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(dev_points);
	hipFree(dev_neighbour);
	hipFree(dev_neighbourcnt);
	hipFree(dev_incidenttet);
	hipFree(dev_incidenttetcnt);
	hipFree(dev_meshtets);
	hipFree(dev_pointgroup);
}

/************************** end of smoothing *******************************/

/************************** Parallel Flipping ******************************/
__global__ void flip23_explore(float *points, int *meshtets, struct cu_flip23face *face, int facecnt, 
							   struct cu_halfface *halfface, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int facepoint[3];
	int toppoint[2];
	int tet[2];
	int tetpoint[8];
	int newtet[12];
	float qualbefore, qualafter;
	//struct cu_flip23face currface;

	while(tid < facecnt)
	{
		//currface = face[tid];

		// check if it is a boundary face
		if (face[tid].hf[0] == -1 || face[tid].hf[1] == -1)
		{
			tid += offset;
			continue;
		}

		// get face points
		facepoint[0] = halfface[face[tid].hf[0]].pointhandle[0];
		facepoint[1] = halfface[face[tid].hf[0]].pointhandle[1];
		facepoint[2] = halfface[face[tid].hf[0]].pointhandle[2];

		// get tet relative data : tet[2]  tetpoint[2][4]  toppoint[2]
		for (int i = 0; i < 2; i ++)
		{
			// get two tets incident to the current face
			tet[i] = face[tid].hf[i]>>2;
			tetpoint[i<<2]     = meshtets[tet[i]<<2];
			tetpoint[(i<<2)+1] = meshtets[(tet[i]<<2)+1];
			tetpoint[(i<<2)+2] = meshtets[(tet[i]<<2)+2];
			tetpoint[(i<<2)+3] = meshtets[(tet[i]<<2)+3];

			for (int j = 0; j < 4; j++)
			{
				if (tetpoint[i*4+j] != facepoint[0] &&
					tetpoint[i*4+j] != facepoint[1] &&
					tetpoint[i*4+j] != facepoint[2])
				{
					toppoint[i] = tetpoint[i*4+j];
					break;
				}
			}
		}

		// get new tets
		newtet[0]  = toppoint[0]; 
		newtet[1]  = facepoint[0];
		newtet[2]  = toppoint[1];
		newtet[3]  = facepoint[2];

		newtet[4]  = toppoint[0];
		newtet[5]  = facepoint[0];
		newtet[6]  = facepoint[1];
		newtet[7]  = toppoint[1];

		newtet[8]  = toppoint[0];
		newtet[9]  = facepoint[1];
		newtet[10] = facepoint[2];
		newtet[11] = toppoint[1];

		// calculate original and new quality
		qualbefore = mintetquality(points, tetpoint, 2, qualmeasure);
		qualafter = mintetquality(points, newtet, 3, qualmeasure);

		// set face value
		face[tid].quality = qualbefore;
		face[tid].val = qualafter - qualbefore;

		tid += offset;
	}
}


__global__ void flip23 (int *meshtets, int tetcnt, struct cu_flip23face *face, int facecnt, struct cu_halfface *halfface, int *selectface, int selectfacecnt)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int facepoint[3];
	int toppoint[2];
	int tet[2];
	int tetpoint[8];
	int pidx;
	int tetpidx[2][5];   // record the tetra inner index of five points
	int newtet;
	cu_flip23face currface;
	int faceset[9];
	int i, j, k;

	while(tid < selectfacecnt)
	{
		currface = face[selectface[tid]];
		faceset[0] = selectface[tid];

		// get face points
		facepoint[0] = halfface[currface.hf[0]].pointhandle[0];
		facepoint[1] = halfface[currface.hf[0]].pointhandle[1];
		facepoint[2] = halfface[currface.hf[0]].pointhandle[2];

		// get tet relative data : tet[2]  tetpoint[2][4]  toppoint[2]
		for (i = 0; i < 2; i ++)
		{
			// get two tets incident to the current face
			tet[i] = currface.hf[i]>>2;
			tetpoint[i<<2]     = meshtets[tet[i]<<2];
			tetpoint[(i<<2)+1] = meshtets[(tet[i]<<2)+1];
			tetpoint[(i<<2)+2] = meshtets[(tet[i]<<2)+2];
			tetpoint[(i<<2)+3] = meshtets[(tet[i]<<2)+3];

			for (j = 0; j < 4; j++)
			{
				for (k = 0; k < 3; k++)
				{
					if (tetpoint[(i<<2)+j] == facepoint[k])
					{
						tetpidx[i][2+k] = j;
						break;
					}
				}

				if (tetpoint[i*4+j] != facepoint[0] &&
					tetpoint[i*4+j] != facepoint[1] &&
					tetpoint[i*4+j] != facepoint[2])
				{
					toppoint[i] = tetpoint[i*4+j];
					tetpidx[i][i] = j;
				}
			}
		}

		faceset[1] = halfface[(tet[0]<<2)+tetpidx[0][2]].face;
		faceset[2] = halfface[(tet[0]<<2)+tetpidx[0][3]].face;
		faceset[3] = halfface[(tet[0]<<2)+tetpidx[0][4]].face;
		faceset[4] = halfface[(tet[1]<<2)+tetpidx[1][2]].face;
		faceset[5] = halfface[(tet[1]<<2)+tetpidx[1][3]].face;
		faceset[6] = halfface[(tet[1]<<2)+tetpidx[1][4]].face;

		/* Set new tetras */
		// new tet0
		pidx = tet[0]<<2;
		meshtets[pidx]   = toppoint[0];   meshtets[pidx+1] = facepoint[0];
		meshtets[pidx+2] = toppoint[1];   meshtets[pidx+3] = facepoint[2];

		halfface[pidx].pointhandle[0] = meshtets[pidx+1];
		halfface[pidx].pointhandle[1] = meshtets[pidx+2];
		halfface[pidx].pointhandle[2] = meshtets[pidx+3];

		halfface[pidx+1].pointhandle[0] = meshtets[pidx];
		halfface[pidx+1].pointhandle[1] = meshtets[pidx+3];
		halfface[pidx+1].pointhandle[2] = meshtets[pidx+2];

		halfface[pidx+2].pointhandle[0] = meshtets[pidx];
		halfface[pidx+2].pointhandle[1] = meshtets[pidx+1];
		halfface[pidx+2].pointhandle[2] = meshtets[pidx+3];

		halfface[pidx+3].pointhandle[0] = meshtets[pidx];
		halfface[pidx+3].pointhandle[1] = meshtets[pidx+2];
		halfface[pidx+3].pointhandle[2] = meshtets[pidx+1];

		// new tet1
		pidx = tet[1]<<2;
		meshtets[pidx]   = toppoint[0];   meshtets[pidx+1] = facepoint[0];
		meshtets[pidx+2] = facepoint[1];  meshtets[pidx+3] = toppoint[1];

		halfface[pidx].pointhandle[0] = meshtets[pidx+1];
		halfface[pidx].pointhandle[1] = meshtets[pidx+2];
		halfface[pidx].pointhandle[2] = meshtets[pidx+3];

		halfface[pidx+1].pointhandle[0] = meshtets[pidx];
		halfface[pidx+1].pointhandle[1] = meshtets[pidx+3];
		halfface[pidx+1].pointhandle[2] = meshtets[pidx+2];

		halfface[pidx+2].pointhandle[0] = meshtets[pidx];
		halfface[pidx+2].pointhandle[1] = meshtets[pidx+1];
		halfface[pidx+2].pointhandle[2] = meshtets[pidx+3];

		halfface[pidx+3].pointhandle[0] = meshtets[pidx];
		halfface[pidx+3].pointhandle[1] = meshtets[pidx+2];
		halfface[pidx+3].pointhandle[2] = meshtets[pidx+1];

		// add new tet
		newtet = tid+tetcnt;
		pidx = newtet<<2;
		meshtets[pidx]   = toppoint[0];  meshtets[pidx+1] = facepoint[1];
		meshtets[pidx+2] = facepoint[2]; meshtets[pidx+3] = toppoint[1];

		halfface[pidx].pointhandle[0] = meshtets[pidx+1];
		halfface[pidx].pointhandle[1] = meshtets[pidx+2];
		halfface[pidx].pointhandle[2] = meshtets[pidx+3];

		halfface[pidx+1].pointhandle[0] = meshtets[pidx];
		halfface[pidx+1].pointhandle[1] = meshtets[pidx+3];
		halfface[pidx+1].pointhandle[2] = meshtets[pidx+2];

		halfface[pidx+2].pointhandle[0] = meshtets[pidx];
		halfface[pidx+2].pointhandle[1] = meshtets[pidx+1];
		halfface[pidx+2].pointhandle[2] = meshtets[pidx+3];

		halfface[pidx+3].pointhandle[0] = meshtets[pidx];
		halfface[pidx+3].pointhandle[1] = meshtets[pidx+2];
		halfface[pidx+3].pointhandle[2] = meshtets[pidx+1];

		// add two new face
		face[(facecnt+tid)*2].hf[0] = 4*tet[0]+1;
		face[(facecnt+tid)*2].hf[1] = 4*newtet+1;
		face[(facecnt+tid)*2+1].hf[0] = 4*tet[1]+1;
		face[(facecnt+tid)*2+1].hf[1] = 4*newtet+2;

		faceset[7] = (facecnt+tid)*2;
		faceset[8] = (facecnt+tid)*2+1;

		halfface[4*tet[0]+1].face = (facecnt+tid)*2;
		halfface[4*newtet+1].face = (facecnt+tid)*2;
		halfface[4*tet[1]+1].face = (facecnt+tid)*2+1;
		halfface[4*newtet+2].face = (facecnt+tid)*2+1;

		// update some old face and halfface relationship
		face[selectface[tid]].hf[0] = 4*tet[1]+2;
		face[selectface[tid]].hf[1] = 4*tet[0]+3;
		halfface[4*tet[1]+2].face = selectface[tid];
		halfface[4*tet[0]+3].face = selectface[tid];

		// update new tet0
		halfface[4*tet[0]].face = faceset[5];
		if (face[faceset[5]].hf[0]>>2 == tet[1])
			face[faceset[5]].hf[0] = 4*tet[0];
		else
			face[faceset[5]].hf[1] = 4*tet[0];

		halfface[4*tet[0]+2].face = faceset[2];
		if (face[faceset[2]].hf[0]>>2 == tet[0])
			face[faceset[2]].hf[0] = 4*tet[0]+2;
		else
			face[faceset[2]].hf[1] = 4*tet[0]+2;

		// update new tet1
		halfface[4*tet[1]].face = faceset[6];
		if (face[faceset[6]].hf[0]>>2 == tet[1])
			face[faceset[6]].hf[0] = 4*tet[1];
		else
			face[faceset[6]].hf[1] = 4*tet[1];

		halfface[4*tet[1]+3].face = faceset[3];
		if (face[faceset[3]].hf[0]>>2 == tet[0])
			face[faceset[3]].hf[0] = 4*tet[1]+3;
		else
			face[faceset[3]].hf[1] = 4*tet[1]+3;

		// update newtet
		halfface[4*newtet].face = faceset[4];
		if (face[faceset[4]].hf[0]>>2 == tet[1])
			face[faceset[4]].hf[0] = 4*newtet;
		else
			face[faceset[4]].hf[1] = 4*newtet;

		halfface[4*newtet+3].face = faceset[1];
		if (face[faceset[1]].hf[0]>>2 == tet[0])
			face[faceset[1]].hf[0] = 4*newtet + 3;
		else
			face[faceset[1]].hf[1] = 4*newtet + 3;

		// add offset and go next loop
		tid += offset;
	}
}

void faceSelecting(struct cu_flip23face *face, int facecnt, int tetcnt, int* &newface, int &newfacecnt)
{
	int selectfacecnt = 0;
	int* selectface = new int[facecnt];
	int i, j;
	float threashold = float(1.0e-5);

	/* pick out the faces by flipping succeed val
	and sort it*/
	for (i = 0; i < facecnt; i++)
	{
		if (!(face[i].val > threashold))
			continue;

		// insert the face
		for (j = selectfacecnt-1; j > -1 ; j--)
		{
			if (face[selectface[j]].quality > face[i].quality)
				selectface[j+1] = selectface[j];
			else 
				break;
		}
		selectface[j+1] = i;
		++ selectfacecnt;
	}

	/* if there are no any face meeting the requirement */
	if (!selectfacecnt)
	{
		newfacecnt = 0;
		newface = NULL;
		return;
	}

	/* select a new face set in which any two of them not in a same tet */
	int tet[2];
	bool *tetflag;
	tetflag = new bool[tetcnt];
	memset(tetflag, 1, tetcnt*sizeof(bool));

	// push the first face
	newface = new int[facecnt];
	newface[0] = selectface[0];
	newfacecnt = 1;

	// set flags of tets incident to the face
	tetflag[(face[newface[0]].hf[0])>>2] = 0;
	tetflag[(face[newface[0]].hf[1])>>2] = 0;

	/* if the tets incident to a face are available, 
	then add the face into the array*/
	for (i = 1; i < selectfacecnt; i++)
	{
		tet[0] = (face[selectface[i]].hf[0])>>2;
		tet[1] = (face[selectface[i]].hf[1])>>2;
		if (tetflag[tet[0]] && tetflag[tet[1]])
		{
			newface[newfacecnt++] = selectface[i];
			tetflag[tet[0]] = 0;
			tetflag[tet[1]] = 0;
		}
	}
}

extern "C" void cuda_flip23(float *points, int pointcnt, int *meshtets_, int tetcnt, int *face, int facecnt, 
							int *halfface, int halffacecnt, int qualmeasure, float& qualbefore_, float& qualafter_, 
							int &flipsucc, float &time)
{
	// create face and halfface structure
	int *meshtets, *tempmeshtets;
	struct cu_flip23face *cface, *tempcafce;
	struct cu_halfface *chalfface, *tempchalfface;
	float qualbefore;
	float qualafter;
	int tetcapacity;
	int facecapacity;
	int halffacecapacity;

	tetcapacity = int(1.1*tetcnt);
	facecapacity = int(1.1*facecnt);
	halffacecapacity = int(1.1*halffacecnt);
	meshtets = new int[4*tetcapacity];
	cface = new struct cu_flip23face[facecapacity];
	chalfface = new struct cu_halfface[halffacecapacity];

	memcpy(meshtets, meshtets_, 4*tetcnt*sizeof(int));

	// calculate quality before flipping
	cuda_tetquality(points, pointcnt, meshtets, tetcnt, qualmeasure, qualbefore);

	// CUDA Parallel
	// �����豸�洢�ռ�
	int loop = 1;
	float *dev_points;
	int *dev_meshtets, *dev_tempmeshtets;
	struct cu_flip23face *dev_face, *dev_tempface;
	struct cu_halfface *dev_halfface, *dev_temphalfface;
	int *selectface;
	int selectfacecnt;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int i = 0; i < facecnt; i++)
	{
		cface[i].hf[0] = face[2*i];
		cface[i].hf[1] = face[2*i+1];
		cface[i].quality = -1;
		cface[i].val = -1;
	}

	for (int i = 0; i < halffacecnt; i++)
	{
		chalfface[i].pointhandle[0] = halfface[4*i];
		chalfface[i].pointhandle[1] = halfface[4*i+1];
		chalfface[i].pointhandle[2] = halfface[4*i+2];
		chalfface[i].face = halfface[4*i+3];
	}


	// flip23 �������µ������塢������棬�����Ԥ����һ���ֿռ�
	hipMalloc((void**)&dev_points, 3*pointcnt*sizeof(float));
	hipMalloc((void**)&dev_meshtets, 4*tetcapacity*sizeof(int));
	hipMalloc((void**)&dev_face, facecapacity*sizeof(cu_flip23face));
	hipMalloc((void**)&dev_halfface, halffacecapacity*sizeof(cu_halfface));

	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_meshtets, meshtets, 4*tetcnt*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_face, cface, facecnt*sizeof(cu_flip23face), hipMemcpyHostToDevice);
	hipMemcpy(dev_halfface, chalfface, halffacecnt*sizeof(cu_halfface), hipMemcpyHostToDevice);

	while(loop)
	{
		// ���ú˺���
		int blocks = imin(pointcnt, BlockPerGrid);

		/* strategy : flip test -> select proper face set -> do flipping */
		// step1 : flip test
		flip23_explore<<<blocks, (facecnt+blocks-1)/blocks>>>(dev_points, dev_meshtets, dev_face, facecnt, dev_halfface, qualmeasure);
		hipMemcpy(cface, dev_face, facecnt*sizeof(cu_flip23face), hipMemcpyDeviceToHost);

		// step2 : select proper face set
		faceSelecting(cface, facecnt, tetcnt, selectface, selectfacecnt);

		// step3 : do flipping
		if (selectfacecnt)
		{
			// check if the memory is not enough, apply more
			if (tetcnt + selectfacecnt > tetcapacity)
			{
				tetcapacity = int(1.2*tetcapacity);
				hipMalloc((void**)&dev_tempmeshtets, 4*tetcnt*sizeof(int));
				hipMemcpy(dev_tempmeshtets, dev_meshtets, 4*tetcnt*sizeof(int), hipMemcpyDeviceToDevice);
				hipFree(dev_meshtets);
				hipMalloc((void**)&dev_meshtets, 4*tetcapacity*sizeof(int));
				hipMemcpy(dev_meshtets, dev_tempmeshtets, 4*tetcnt*sizeof(int), hipMemcpyDeviceToDevice);
				hipFree(dev_tempmeshtets);

				tempmeshtets = new int[4*tetcnt];
				memcpy(tempmeshtets, meshtets, 4*tetcnt*sizeof(int));
				delete [] meshtets;
				meshtets = new int[4*tetcapacity];
				memcpy(meshtets, tempmeshtets, 4*tetcnt*sizeof(int));
				delete [] tempmeshtets;

				halffacecapacity = int(1.2*halffacecapacity);
				hipMalloc((void**)&dev_temphalfface, halffacecnt*sizeof(struct cu_halfface));
				hipMemcpy(dev_temphalfface, dev_halfface, halffacecnt*sizeof(struct cu_halfface), hipMemcpyDeviceToDevice);
				hipFree(dev_halfface);
				hipMalloc((void**)&dev_halfface, halffacecapacity*sizeof(struct cu_halfface));
				hipMemcpy(dev_halfface, dev_temphalfface, halffacecnt*sizeof(struct cu_halfface), hipMemcpyDeviceToDevice);
				hipFree(dev_temphalfface);

				tempchalfface = new struct cu_halfface[halffacecnt];
				memcpy(tempchalfface, chalfface, halffacecnt*sizeof(cu_halfface));
				delete [] chalfface;
				chalfface = new struct cu_halfface[halffacecapacity];
				memcpy(chalfface, tempchalfface, halffacecnt*sizeof(cu_halfface));
				delete [] tempchalfface;
			}
			if (facecnt + 3*selectfacecnt > facecapacity)
			{
				facecapacity = int(1.2*facecapacity);
				hipMalloc((void**)&dev_tempface, facecnt*sizeof(struct cu_flip23face));
				hipMemcpy(dev_tempface, dev_face, facecnt*sizeof(struct cu_flip23face), hipMemcpyDeviceToDevice);
				hipFree(dev_face);
				hipMalloc((void**)&dev_face, facecapacity*sizeof(struct cu_flip23face));
				hipMemcpy(dev_face, dev_tempface, facecnt*sizeof(struct cu_flip23face), hipMemcpyDeviceToDevice);
				hipFree(dev_tempface);

				tempcafce = new struct cu_flip23face[facecnt];
				memcpy(tempcafce, cface, facecnt*sizeof(struct cu_flip23face));
				delete [] cface;
				cface = new struct cu_flip23face[facecapacity];
				memcpy(cface, tempcafce, facecnt*sizeof(struct cu_flip23face));
				delete [] tempcafce;
			}
 
			int *dev_selectface;
			hipMalloc((void**)&dev_selectface, selectfacecnt*sizeof(int));
			hipMemcpy(dev_selectface, selectface, selectfacecnt*sizeof(int), hipMemcpyHostToDevice);
			flip23<<<blocks, (selectfacecnt+blocks-1)/blocks>>>(dev_meshtets, tetcnt, dev_face, facecnt, dev_halfface, dev_selectface, selectfacecnt);

			tetcnt += selectfacecnt;
			facecnt += 2*selectfacecnt;
			halffacecnt += 4*selectfacecnt;

			hipMemcpy(meshtets, dev_meshtets, 4*tetcnt*sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_selectface);
		}
		/* end of strategy2 */

		// malloc in faceSelecting function
		delete [] selectface;
		--loop;
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elaspsedTime;
	hipEventElapsedTime(&elaspsedTime, start, stop);

	time = elaspsedTime;

	// calculate quality after flipping
	qualafter = 1.0;
	hipMemcpy(meshtets, dev_meshtets, 4*tetcnt*sizeof(int), hipMemcpyDeviceToHost);
	cuda_tetquality(points, pointcnt, meshtets, tetcnt, qualmeasure, qualafter);

	qualbefore_ = qualbefore;
	qualafter_ = qualafter;
	flipsucc = selectfacecnt;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_points);
	hipFree(dev_meshtets);
	hipFree(dev_halfface);
	hipFree(dev_face);

	delete [] cface;
	delete [] chalfface;
}

__global__ void flip32_explore(float *points, int *meshtets, struct cu_flip32edge *edge, int edgecnt, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int tet[12];
	int newtet[8];
	int pidxvec[5];
	//int idx;
	int i,j;
	//int tetidx;
	//int pidx;
	float qualbefore, qualafter;
	//cu_flip32edge curredge;

	while(tid < edgecnt)
	{
		//curredge = edge[tid];

		/* fetch local data */
		// edge incident tets
		for (i = 0; i < 3; i ++)
		{
			//tetidx = edge[tid].tet[i];
			for (j = 0; j < 4; j++)
				tet[i*4+j] = meshtets[edge[tid].tet[i]*4+j];
		}

		/* flip32 incident five points: 
		   two are the endpoints of edge, 
		   the other three have to get from incident tets*/
		pidxvec[0] = edge[tid].p[0];
		pidxvec[1] = edge[tid].p[1];

		j = 2;
		// get two other points from one tet
		for (i = 0; i < 4; i++)
		{
			//pidx = tet[i];
			if (tet[i] != pidxvec[0] && tet[i] != pidxvec[1])
			{
				pidxvec[j] = tet[i];
				++j;
			}
		}

		// wrong data
		if (j != 4)
		{
			tid += offset;
			continue;
		}

		// get the last one from another tet
		pidxvec[4] = -1;
		for (i = 4; i < 12; i++)
		{
			//pidx = tet[i+4];
			if (tet[i] != pidxvec[0] && tet[i] != pidxvec[1] &&
				tet[i] != pidxvec[2] && tet[i] != pidxvec[3])
			{
				pidxvec[4] = tet[i];
				break;
			}
		}

		if (pidxvec[4] == -1)
		{
			tid += offset;
			continue;
		}
		/* end of data fetch */

		// calculate quality before flip32
		qualbefore = mintetquality(points, tet, 3, qualmeasure);

		// get new tets : the five points are not in order, so we probably try twice
		// conbination 1
		newtet[0] = pidxvec[0];
		newtet[1] = pidxvec[2];
		newtet[2] = pidxvec[3];
		newtet[3] = pidxvec[4];

		newtet[4] = pidxvec[1];
		newtet[5] = pidxvec[2];
		newtet[6] = pidxvec[4];
		newtet[7] = pidxvec[3];

		edge[tid].order = 0;

		qualafter = mintetquality(points, newtet, 2, qualmeasure);

		// if generate a reverse tet, try another way
		if (qualafter < 0)
		{
			newtet[0] = pidxvec[0];
			newtet[1] = pidxvec[2];
			newtet[2] = pidxvec[4];
			newtet[3] = pidxvec[3];

			newtet[4] = pidxvec[1];
			newtet[5] = pidxvec[2];
			newtet[6] = pidxvec[3];
			newtet[7] = pidxvec[4];

			edge[tid].order = 1;

			qualafter = mintetquality(points, newtet, 2, qualmeasure);
		}

		// set data
		edge[tid].quality = qualbefore;
		edge[tid].val = qualafter - qualbefore;

		tid += offset;
	}
}

__global__ void flip32(float *points, int *meshtets, struct cu_flip32edge *edge, 
					   int *selectedge, int selectedgecnt, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int tet[12];
	int newtet[8];
	int pidxvec[5];
	int i, j;
	//int pidx;
	//float qual;
	cu_flip32edge curredge;

	while(tid < selectedgecnt)
	{
		curredge = edge[selectedge[tid]];

		/* fetch local data */
		// edge incident tets
		for (i = 0; i < 3; i ++)
		{
			for (j = 0; j < 4; j++)
				tet[i*4+j] = meshtets[curredge.tet[i]*4+j];
		}

		/* flip32 incident five points: 
		   two are the endpoints of edge, 
		   the other three have to get from incident tets*/
		pidxvec[0] = curredge.p[0];
		pidxvec[1] = curredge.p[1];

		j = 2;
		// get two other points from one tet
		for (i = 0; i < 4; i++)
		{
			//pidx = tet[i];
			if (tet[i] != pidxvec[0] && tet[i] != pidxvec[1])
			{
				pidxvec[j] = tet[i];
				++j;
			}
		}

		// wrong data
		if (j != 4)
		{
			tid += offset;
			continue;
		}

		// get the last one from another tet
		pidxvec[4] = -1;
		for (i = 4; i < 12; i++)
		{
			//pidx = tet[i+4];
			if (tet[i] != pidxvec[0] && tet[i] != pidxvec[1] &&
				tet[i] != pidxvec[2] && tet[i] != pidxvec[3])
			{
				pidxvec[4] = tet[i];
				break;
			}
		}

		if (pidxvec[4] == -1)
		{
			tid += offset;
			continue;
		}
		/* end of data fetch */

		// calculate quality before flip32
		//qualbefore = mintetquality(points, tet, 3, qualmeasure);

		// get new tets : the five points are not in order, so we probably try twice
		// conbination 1
		if (curredge.order == 0)
		{
			newtet[0] = pidxvec[0];
			newtet[1] = pidxvec[2];
			newtet[2] = pidxvec[3];
			newtet[3] = pidxvec[4];

			newtet[4] = pidxvec[1];
			newtet[5] = pidxvec[2];
			newtet[6] = pidxvec[4];
			newtet[7] = pidxvec[3];
		}
		else
		{
			newtet[0] = pidxvec[0];
			newtet[1] = pidxvec[2];
			newtet[2] = pidxvec[4];
			newtet[3] = pidxvec[3];

			newtet[4] = pidxvec[1];
			newtet[5] = pidxvec[2];
			newtet[6] = pidxvec[3];
			newtet[7] = pidxvec[4];
		}


		// update data
		//if (qualafter - qualbefore > 1.0e-5)
		//{
			for (i = 0; i < 2; i++)
			{
				for (j = 0; j < 4; j++)
				{
					meshtets[curredge.tet[i]*4+j] = newtet[i*4+j];
				}
			}

			for (i = 0; i < 4; i++)
			{
				meshtets[curredge.tet[2]*4+i] = -1;
			}
		//}
		tid += offset;
	}
}


void edgeSelecting(cu_flip32edge *cuedge, int edgecnt, int tetcnt, int *selectedge, int &selectedgecnt)
{
	int sedgecnt = 0;
	int* sedge = new int[edgecnt];
	int i, j;
	float threashold = float(1.0e-5);

	/* pick out the faces by flipping succeed val
	and sort it*/
	for (i = 0; i < edgecnt; i++)
	{
		if (!(cuedge[i].val > threashold))
			continue;

		// insert the face
		for (j = sedgecnt-1; j > -1 ; j--)
		{
			if (cuedge[sedge[j]].quality > cuedge[i].quality)
				sedge[j+1] = sedge[j];
			else 
				break;
		}
		sedge[j+1] = i;
		++ sedgecnt;
	}

	/* if there are no any face meeting the requirement */
	if (!sedgecnt)
	{
		selectedgecnt = 0;
		selectedge = NULL;
		return;
	}

	/* select a new face set in which any two of them not in a same tet */
	int tet[3];
	bool *tetflag;
	tetflag = new bool[tetcnt];
	memset(tetflag, 1, tetcnt*sizeof(bool));

	// push the first face
	selectedge[0] = sedge[0];
	selectedgecnt = 1;

	// set flags of tets incident to the face
	tetflag[cuedge[selectedge[0]].tet[0]] = 0;
	tetflag[cuedge[selectedge[0]].tet[1]] = 0;
	tetflag[cuedge[selectedge[0]].tet[2]] = 0;

	/* if the tets incident to a face are available, 
	then add the face into the array*/
	for (i = 1; i < sedgecnt; i++)
	{
		tet[0] = cuedge[sedge[i]].tet[0];
		tet[1] = cuedge[sedge[i]].tet[1];
		tet[2] = cuedge[sedge[i]].tet[2];
		if (tetflag[tet[0]] && tetflag[tet[1]] && tetflag[tet[2]])
		{
			selectedge[selectedgecnt++] = sedge[i];
			tetflag[tet[0]] = 0;
			tetflag[tet[1]] = 0;
			tetflag[tet[2]] = 0;
		}
	}
}

extern "C" void cuda_flip32(float *points, int pointcnt, int *meshtets, int tetcnt, int *flipedge, int edgecnt, 
				            int qualmeasure, float& qualbefore_, float& qualafter_, int &flipsucc, float &time)
{
	

	float qualbefore, qualafter;
	float *dev_points;
	int *dev_meshtets;
	struct cu_flip32edge *dev_edge;
	int *selectedge;
	int selectedgecnt;
	int loop = 4;

	// calculate quality before flip32
	cuda_tetquality(points, pointcnt, meshtets, tetcnt, qualmeasure, qualbefore);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	struct cu_flip32edge *cuedge;
	cuedge = new struct cu_flip32edge[edgecnt];

	for (int i = 0; i < edgecnt; i++)
	{
		cuedge[i].p[0] = flipedge[i*5];
		cuedge[i].p[1] = flipedge[i*5+1];
		cuedge[i].tet[0] = flipedge[i*5+2];
		cuedge[i].tet[1] = flipedge[i*5+3];
		cuedge[i].tet[2] = flipedge[i*5+4];
		cuedge[i].quality = 1.0;
		cuedge[i].val = -1.0;
	}

	hipMalloc((void**)&dev_points, 3*pointcnt*sizeof(float));
	hipMalloc((void**)&dev_meshtets, 4*tetcnt*sizeof(int));
	hipMalloc((void**)&dev_edge, edgecnt*sizeof(cu_flip32edge));

	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_meshtets, meshtets, 4*tetcnt*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_edge, cuedge, edgecnt*sizeof(cu_flip32edge), hipMemcpyHostToDevice);
	
	flipsucc = 0;
	//while(loop)
	{
		/***********do parallel flip32**********/
		// flip32 explore
		int blocks = imin(edgecnt, BlockPerGrid);
		int threads = imin((edgecnt+blocks-1)/blocks, ThreadPerBlock);
		flip32_explore<<<blocks, threads>>>(dev_points, dev_meshtets, dev_edge, edgecnt, qualmeasure);
		hipMemcpy(cuedge, dev_edge, edgecnt*sizeof(cu_flip32edge), hipMemcpyDeviceToHost);

		// edge selecting
		selectedge = new int[edgecnt];
		edgeSelecting(cuedge, edgecnt, tetcnt, selectedge, selectedgecnt);

		// flip32 (set -1 to the meshtets[] of invalid tets)
		if (selectedgecnt)
		{
			int *dev_selectedge;
			hipMalloc((void**)&dev_selectedge, selectedgecnt*sizeof(int));
			hipMemcpy(dev_selectedge, selectedge, selectedgecnt*sizeof(int), hipMemcpyHostToDevice);

			// do flip32
			blocks = imin(selectedgecnt, BlockPerGrid);
			flip32<<<blocks, (selectedgecnt+blocks-1)/blocks>>>(dev_points, dev_meshtets, dev_edge, dev_selectedge, selectedgecnt, qualmeasure);

			flipsucc += selectedgecnt;
			hipFree(dev_selectedge);
		}
		--loop;
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elaspsedTime;
	hipEventElapsedTime(&elaspsedTime, start, stop);

	time = elaspsedTime;

	// calculate quality after flipping
	hipMemcpy(meshtets, dev_meshtets, 4*tetcnt*sizeof(int), hipMemcpyDeviceToHost);
	cuda_tetquality(points, pointcnt, meshtets, tetcnt, qualmeasure, qualafter);

	qualbefore_ = qualbefore;
	qualafter_ = qualafter;

	delete [] selectedge;  // malloc in edge selecting function
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_points);
	hipFree(dev_meshtets);
}

__global__ void flip32_explore_new(float *points, int *meshtets, cu_edge *edge, int edgecnt, cu_halfedge *halfedge, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int tet[12];
	int newtet[8];
	int pidxvec[5];
	int idx;
	int tetidx;
	int pidx;
	float qualbefore, qualafter;
	cu_edge curredge;

	while(tid < edgecnt)
	{
		curredge = edge[tid];

		// if the edge is boundary or the amount of incident tetras is not 3, go next
		if (curredge.is_boundary || curredge.halfedgecnt != 6)
		{
			tid += offset;
			continue;
		}

		/* fetch local data */
		// edge incident tets
		for (int i = 0; i < 3; i ++)
		{
			tetidx = curredge.halfedge[i*2]>>2;
			for (int j = 0; j < 4; j++)
				tet[i*4+j] = meshtets[tetidx*4+j];
		}

		/* flip32 incident five points: 
		   two are the endpoints of edge, 
		   the other three have to get from incident tets*/
		pidxvec[0] = halfedge[curredge.halfedge[0]].fromv;
		pidxvec[1] = halfedge[curredge.halfedge[0]].tov;

		idx = 0;
		// get two other points from one tet
		for (int i = 0; i < 4; i++)
		{
			pidx = tet[i];
			if (pidx != pidxvec[0] && pidx != pidxvec[1])
			{
				pidxvec[idx+2] = pidx;
				++idx;
			}
		}

		// wrong data
		if (idx != 2)
		{
			tid += offset;
			continue;
		}

		// get the last one from another tet
		for (int i = 0; i < 4; i++)
		{
			pidx = tet[i+4];
			if (pidx != pidxvec[0] && pidx != pidxvec[1] &&
				pidx != pidxvec[2] && pidx != pidxvec[3])
			{
				pidxvec[4] = pidx;
				break;
			}
		}
		/* end of data fetch */

		// calculate quality before flip32
		qualbefore = mintetquality(points, tet, 3, qualmeasure);

		// get new tets : the five points are not in order, so we probably try twice
		// conbination 1
		newtet[0] = pidxvec[0];
		newtet[1] = pidxvec[2];
		newtet[2] = pidxvec[3];
		newtet[3] = pidxvec[4];

		newtet[4] = pidxvec[1];
		newtet[5] = pidxvec[2];
		newtet[6] = pidxvec[4];
		newtet[7] = pidxvec[3];

		qualafter = mintetquality(points, newtet, 2, qualmeasure);

		// if generate a reverse tet, try another way
		if (qualafter < 0)
		{
			newtet[0] = pidxvec[0];
			newtet[1] = pidxvec[2];
			newtet[2] = pidxvec[4];
			newtet[3] = pidxvec[3];

			newtet[4] = pidxvec[1];
			newtet[5] = pidxvec[2];
			newtet[6] = pidxvec[3];
			newtet[7] = pidxvec[4];

			qualafter = mintetquality(points, newtet, 2, qualmeasure);
		}

		// set data
		edge[tid].quality = qualbefore;
		edge[tid].val = qualafter - qualbefore;

		tid += offset;
	}
}

void edgeSelecting_new(cu_edge *cuedge, int edgecnt, int tetcnt, int *selectedge, int &selectedgecnt)
{
	int sedgecnt = 0;
	int* sedge = new int[edgecnt];
	int i, j;
	float threashold = MINIMPROVEMENT;

	/* pick out the faces by flipping succeed val
	and sort it*/
	for (i = 0; i < edgecnt; i++)
	{
		if (!(cuedge[i].val > threashold))
			continue;

		// insert the face
		for (j = sedgecnt-1; j > -1 ; j--)
		{
			if (cuedge[sedge[j]].quality > cuedge[i].quality)
				sedge[j+1] = sedge[j];
			else 
				break;
		}
		sedge[j+1] = i;
		++ sedgecnt;
	}

	/* if there are no any face meeting the requirement */
	if (!sedgecnt)
	{
		selectedgecnt = 0;
		selectedge = NULL;
		return;
	}

	/* select a new face set in which any two of them not in a same tet */
	int tet[3];
	bool *tetflag;
	tetflag = new bool[tetcnt];
	memset(tetflag, 1, tetcnt*sizeof(bool));

	// push the first face
	selectedge[0] = sedge[0];
	selectedgecnt = 1;

	// set flags of tets incident to the face
	tetflag[cuedge[selectedge[0]].halfedge[0]>>2] = 0;
	tetflag[cuedge[selectedge[0]].halfedge[2]>>2] = 0;
	tetflag[cuedge[selectedge[0]].halfedge[4]>>2] = 0;

	/* if the tets incident to a face are available, 
	then add the face into the array*/
	for (i = 1; i < sedgecnt; i++)
	{
		tet[0] = cuedge[sedge[i]].halfedge[0]>>2;
		tet[1] = cuedge[sedge[i]].halfedge[2]>>2;
		tet[2] = cuedge[sedge[i]].halfedge[4]>>2;
		if (tetflag[tet[0]] && tetflag[tet[1]] && tetflag[tet[2]])
		{
			selectedge[selectedgecnt++] = sedge[i];
			tetflag[tet[0]] = 0;
			tetflag[tet[1]] = 0;
			tetflag[tet[2]] = 0;
		}
	}
}

__global__ void flip32_new(float *points, int *meshtets, cu_edge *edge, cu_halfedge *halfedge, 
					       int *selectedge, int selectedgecnt, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int tet[12];
	int newtet[8];
	int pidxvec[5];
	int idx;
	int tetidx;
	int pidx;
	float qualbefore, qualafter;
	cu_edge curredge;

	while(tid < selectedgecnt)
	{
		curredge = edge[selectedge[tid]];

		/* fetch local data */
		// edge incident tets
		for (int i = 0; i < 3; i ++)
		{
			tetidx = curredge.halfedge[i*2]>>2;
			for (int j = 0; j < 4; j++)
				tet[i*4+j] = meshtets[tetidx*4+j];
		}

		/* flip32 incident five points: 
		   two are the endpoints of edge, 
		   the other three have to get from incident tets*/
		pidxvec[0] = halfedge[curredge.halfedge[0]].fromv;
		pidxvec[1] = halfedge[curredge.halfedge[0]].tov;

		idx = 0;
		// get two other points from one tet
		for (int i = 0; i < 4; i++)
		{
			pidx = tet[i];
			if (pidx != pidxvec[0] && pidx != pidxvec[1])
			{
				pidxvec[idx+2] = pidx;
				++idx;
			}
		}

		// wrong data
		if (idx != 2)
		{
			tid += offset;
			continue;
		}

		// get the last one from another tet
		for (int i = 0; i < 4; i++)
		{
			pidx = tet[i+4];
			if (pidx != pidxvec[0] && pidx != pidxvec[1] &&
				pidx != pidxvec[2] && pidx != pidxvec[3])
			{
				pidxvec[4] = pidx;
				break;
			}
		}
		/* end of data fetch */

		// calculate quality before flip32
		qualbefore = mintetquality(points, tet, 3, qualmeasure);

		// get new tets : the five points are not in order, so we probably try twice
		// conbination 1
		newtet[0] = pidxvec[0];
		newtet[1] = pidxvec[2];
		newtet[2] = pidxvec[3];
		newtet[3] = pidxvec[4];

		newtet[4] = pidxvec[1];
		newtet[5] = pidxvec[2];
		newtet[6] = pidxvec[4];
		newtet[7] = pidxvec[3];

		qualafter = mintetquality(points, newtet, 2, qualmeasure);

		// if generate a reverse tet, try another way
		if (qualafter < 0)
		{
			newtet[0] = pidxvec[0];
			newtet[1] = pidxvec[2];
			newtet[2] = pidxvec[4];
			newtet[3] = pidxvec[3];

			newtet[4] = pidxvec[1];
			newtet[5] = pidxvec[2];
			newtet[6] = pidxvec[3];
			newtet[7] = pidxvec[4];

			qualafter = mintetquality(points, newtet, 2, qualmeasure);
		}

		// update data
		if (qualafter - qualbefore > 1.0e-5)
		{
			// update tetras
			for (int i = 0; i < 2; i++)
			{
				for (int j = 0; j < 4; j++)
				{
					meshtets[(curredge.halfedge[i*2]>>2)<<2+j] = newtet[i*4+j];

					// update halfface
				}
			}

			for (int i = 0; i < 4; i++)
			{
				meshtets[(curredge.halfedge[4]>>2)<<2+i] = -1;
			}

			// update edges
			curredge.halfedgecnt = 0;  // current edge has been deleted

			//for (int i = 0; i < 3; i++)
			//{
			//	cuPrintf("Tetra %d : ", curredge.tet[i]);
			//	for (int j = 0; j < 4; j++)
			//	{
			//		cuPrintf("%d ", meshtets[curredge.tet[i]*4+j]);
			//	}
			//	cuPrintf("\n");
			//}
			//cuPrintf("\nqualbefore: %f   qualafter: %f \n", qualbefore, qualafter);
		}

		tid += offset;
	}
}

extern "C" void cuda_flip32_new(float *points, int pointcnt, int *meshtets, int tetcnt, int *edge, int edgecnt,
								int *halfedge, int halfedgecnt, int qualmeasure, float &qualbefore_, float &qualafeter_,
								int &flipsucc, float &time)
{
	cu_edge *cedge;
	cu_halfedge *chalfedge;
	cedge = new cu_edge[edgecnt];
	chalfedge = new cu_halfedge[halfedgecnt];

	// build edge
	int idx = 0;
	for (int i = 0; i < edgecnt; i ++)
	{
		cedge[i].is_boundary = edge[idx++];
		cedge[i].halfedgecnt = edge[idx++];
		for (int j = 0; j < cedge[i].halfedgecnt; j++)
		{
			cedge[i].halfedge[j] = edge[idx++];
		}
		cedge[i].quality = cedge[i].val = -1;
	}

	// build halfedge
	for (int i = 0; i < halfedgecnt; i++)
	{
		chalfedge[i].edgehandle = halfedge[i*3];
		chalfedge[i].fromv = halfedge[i*3+1];
		chalfedge[i].tov = halfedge[i*3+2];
	}

	float qualbefore;
	//float qualafter;
	float *dev_points;
	int *dev_meshtets;
	struct cu_edge *dev_edge;
	struct cu_halfedge *dev_halfedge;
	int *selectedge;
	int selectedgecnt;
	int loop = 1;

	// calculate quality before flip32
	cuda_tetquality(points, pointcnt, meshtets, tetcnt, qualmeasure, qualbefore);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&dev_points, 3*pointcnt*sizeof(float));
	hipMalloc((void**)&dev_meshtets, 4*tetcnt*sizeof(int));
	hipMalloc((void**)&dev_edge, edgecnt*sizeof(cu_edge));
	hipMalloc((void**)&dev_halfedge, halfedgecnt*sizeof(cu_halfedge));

	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_meshtets, meshtets, 4*tetcnt*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_edge, cedge, edgecnt*sizeof(cu_edge), hipMemcpyHostToDevice);
	hipMemcpy(dev_halfedge, chalfedge, halfedgecnt*sizeof(cu_halfedge), hipMemcpyHostToDevice);

	flipsucc = 0;
	while(loop)
	{

		/***********do parallel flip32**********/
		// flip32 explore
		int blocks = imin(edgecnt, BlockPerGrid);
		flip32_explore_new<<<blocks, (edgecnt+blocks-1)/blocks>>>(dev_points, dev_meshtets, dev_edge, edgecnt, dev_halfedge, qualmeasure);
		hipMemcpy(cedge, dev_edge, edgecnt*sizeof(cu_edge), hipMemcpyDeviceToHost);

		// edge selecting
		selectedge = new int[edgecnt];
		edgeSelecting_new(cedge, edgecnt, tetcnt, selectedge, selectedgecnt);

		// flip32 (set -1 to the meshtets[] of invalid tets)
		if (selectedgecnt)
		{
			int *dev_selectedge;
			hipMalloc((void**)&dev_selectedge, selectedgecnt*sizeof(int));
			hipMemcpy(dev_selectedge, selectedge, selectedgecnt*sizeof(int), hipMemcpyHostToDevice);

			// do flip32
			blocks = imin(selectedgecnt, BlockPerGrid);
			//flip32<<<blocks, (selectedgecnt+blocks-1)/blocks>>>(dev_points, dev_meshtets, dev_edge, dev_selectedge, selectedgecnt, qualmeasure);

			flipsucc += selectedgecnt;
			hipFree(dev_selectedge);
		}
		-- loop;
	}
}

/************************** end of flipping ******************************/


/************************ New Flip ****************************/
__device__ void add_vec3f(float *a, float *b, float *res)
{
	for (int i = 0; i < 3; i++)
		res[i] = a[i] + b[i];
}

__device__ void minus_vec3f(float *a, float *b, float *res)
{
	for (int i = 0; i < 3; i++)
		res[i] = a[i] - b[i];
}

__device__ float dot_vec3f(float *a, float *b)
{
	return a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
}

__device__ void cross_vec3f(float *a, float *b, float *res)
{
	res[0] = a[1] * b[2] - a[2] * b[1];
	res[1] = a[2] * b[0] - a[0] * b[2];
	res[2] = a[0] * b[1] - a[1] * b[0];
}

__device__ float norm_vec3f(float *a)
{
	return sqrtf(a[0]*a[0]+a[1]*a[1]+a[2]*a[2]);
}

__device__ void normalize_vec3f(float *a)
{
	float norm = norm_vec3f(a);
	for (int i = 0; i < 3; i++)
		a[i] /= norm;
}

__device__ int checkPointInsideTriangle(float triangle[3][3], float p[3])
{
	float edge1[3];
	float edge2[3];
	float faceNormal[3];

	minus_vec3f(triangle[1], triangle[0], edge1);
	minus_vec3f(triangle[2], triangle[0], edge2);
	cross_vec3f(edge1, edge2, faceNormal);
	normalize_vec3f(faceNormal);

	float v[3][3];
	float vec1[3], vec2[3];

	minus_vec3f(triangle[1], triangle[0], vec1);
	minus_vec3f(p, triangle[0], vec2);
	cross_vec3f(vec1, vec2, v[0]);

	minus_vec3f(triangle[2], triangle[1], vec1);
	minus_vec3f(p, triangle[1], vec2);
	cross_vec3f(vec1, vec2, v[1]);

	minus_vec3f(triangle[0], triangle[2], vec1);
	minus_vec3f(p, triangle[2], vec2);
	cross_vec3f(vec1, vec2, v[2]);

	float cosAngle;

	for (int i = 0; i < 3; i ++)
	{
		if (norm_vec3f(v[i]) > 1e-10)
		{
			cosAngle = dot_vec3f(v[i], faceNormal) / norm_vec3f(v[i]);
		}
		else
		{
			cosAngle = 0;
		}
		if (cosAngle < - 1e-10)
		{
			// outside the triangle
			return 0;
		}
		else if (cosAngle == 0)
		{
			// on the triangle boundary
			return -1;
		}
	}
	// inside the triangle
	return 1;
}

__device__ float vertexTriangleDistance(float *points, int triangle[3], int p_, int & pStatus)
{
	float dis;
	dis = 0;

	float trianglev[3][3];
	float edge1[3], edge2[3];
	float faceNormal[3];
	float norm;
	float p[3];
	float pp[3];

	// get p
	for(int i = 0; i < 3; i ++)
		p[i] = points[p_*3+i];

	// get triangle points
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			trianglev[i][j] = points[triangle[i]*3+j];

	minus_vec3f(trianglev[1], trianglev[0], edge1);
	minus_vec3f(trianglev[2], trianglev[0], edge2);
	cross_vec3f(edge1, edge2, faceNormal);
	normalize_vec3f(faceNormal);
	norm = norm_vec3f(faceNormal);
	pStatus = 0;

	dis = (faceNormal[0] * p[0] + faceNormal[1] * p[1] +
		faceNormal[2] * p[2] - faceNormal[0] * trianglev[0][0] -
		faceNormal[1] * trianglev[0][1] - faceNormal[2] * trianglev[0][2]) / norm;

	for(int i = 0; i < 3; i ++)
		pp[i] = p[i] - faceNormal[i]*dis;

	pStatus = checkPointInsideTriangle(trianglev, pp);
	return dis;
}

__device__ bool trianglehasedge(int ph[3], int e[2])
{
	int c = 0;
	for (int i = 0; i < 3; i++)
	{
		if (ph[i] == e[0] || ph[i] == e[1])
			++ c;
	}
	if (c == 2)
		return true;
	return false;
}

__device__ bool trianglehaspoint(int ph[3], int p)
{
	return (ph[0] == p || ph[1] == p || ph[2] == p);
}

/* ȷ�Ϸ�������������Χ��������ֲ��Ƿ񸴺�Ҫ�� */
__device__ bool kitesituationcheck(float *points, cu_tetra *tetra, int tidx, cu_flip23face *face, cu_halfface *halfface, 
								   int e1[2], int e2[2], int qualmeasure)
{
	int hf1[2];
	int hf2[2];
	int ophf1[2];
	int ophf2[2];
	cu_tetra tet1[2];
	cu_tetra tet2[2];
	int tetidx[4];
	int p1[2], p2[2];
	int fidx1, fidx2;
	cu_halfface tmphface;
	cu_flip23face tmpface;

	fidx1 = fidx2 = 0;
	for (int i = 0; i < 4; i ++)
	{
		tmphface = halfface[tidx*4+i];
		if (trianglehasedge(tmphface.pointhandle, e1))
			hf1[fidx1++] = tidx*4+i;
		else if(trianglehasedge(tmphface.pointhandle, e2))
			hf2[fidx2++] = tidx*4+i;
	}

	// find opposite halfface
	tmpface = face[halfface[hf1[0]].face];
	if (tmpface.hf[0] == hf1[0])
		ophf1[0] = tmpface.hf[1];
	else
		ophf1[0] = tmpface.hf[0];

	tmpface = face[halfface[hf1[1]].face];
	if (tmpface.hf[0] == hf1[1])
		ophf1[1] = tmpface.hf[1];
	else
		ophf1[1] = tmpface.hf[0];

	tmpface = face[halfface[hf2[0]].face];
	if (tmpface.hf[0] == hf2[0])
		ophf2[0] = tmpface.hf[1];
	else
		ophf2[0] = tmpface.hf[0];

	tmpface = face[halfface[hf2[1]].face];
	if (tmpface.hf[0] == hf2[1])
		ophf2[1] = tmpface.hf[1];
	else
		ophf2[1] = tmpface.hf[0];

	// get top points
	for (int i = 0; i < 2; i++)
	{
		tet1[i] = tetra[ophf1[i]>>2];
		tetidx[i] = ophf1[i]>>2;
		tmphface = halfface[ophf1[i]];
		for (int j = 0; j < 4; j++)
		{
			if (!trianglehaspoint(tmphface.pointhandle, tet1[i].v[j]))
			{
				p1[i] = tet1[i].v[j];
				break;
			}
		}

		tet2[i] = tetra[ophf2[i]>>2];
		tetidx[i+2] = ophf2[i]>>2;
		tmphface = halfface[ophf2[i]];
		for (int j = 0; j < 4; j++)
		{
			if (!trianglehaspoint(tmphface.pointhandle, tet2[i].v[j]))
			{
				p2[i] = tet2[i].v[j];
				break;
			}
		}
	}

	if (p1[0] == p1[1] && p2[0] == p2[1])
	{
		float p[4][3];
		cu_tetra currtet;
		int newtet[16];
		float qualtet1[2], qualtet2[2], tetqual;
		float qualnewtet[2];
		float minqualbefore, minqualafter1, minqualafter2;

		// calculate quality before newflip
		minqualbefore = 1.0;
		for (int k = 0; k < 2; k ++)
		{
			for (int i = 0; i < 4; i++)
				for (int j = 0; j < 3; j++)
					p[i][j] = points[tet1[k].v[i]*3+j];
			qualtet1[k] = tetquality(p, qualmeasure);
			if (minqualbefore > qualtet1[k])
				minqualbefore = qualtet1[k];

			for (int i = 0; i < 4; i++)
				for (int j = 0; j < 3; j++)
					p[i][j] = points[tet2[k].v[i]*3+j];
			qualtet2[k] = tetquality(p, qualmeasure);
			if (minqualbefore > qualtet2[k])
				minqualbefore = qualtet2[k];
		}

		currtet = tetra[tidx];
		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 3; j++)
				p[i][j] = points[currtet.v[i]*3+j];

		tetqual = tetquality(p, qualmeasure);
		if (minqualbefore > tetqual)
			minqualbefore = tetqual;

		// update tetras' information
		tetra[tidx].quality = minqualbefore;
		tetra[tidx].fliptype = 1;
		tetra[tidx].newflipface[0] = hf1[0];
		tetra[tidx].newflipface[1] = hf1[1];
		tetra[tidx].newflipface[2] = hf2[0];
		tetra[tidx].newflipface[3] = hf2[1];
		tetra[tidx].tet[0] = tetidx[0];
		tetra[tidx].tet[1] = tetidx[1];
		tetra[tidx].tet[2] = tetidx[2];
		tetra[tidx].tet[3] = tetidx[3];
		tetra[tidx].flippoint[0] = p1[0];
		tetra[tidx].flippoint[1] = p2[0];

		// ȷ��flip����
		// ��ϲ���һ��e1��Χ����������+�µ�����������
		newtet[0] = p2[0];
		newtet[1] = halfface[hf1[0]].pointhandle[0];
		newtet[2] = halfface[hf1[0]].pointhandle[1];
		newtet[3] = halfface[hf1[0]].pointhandle[2];

		newtet[4] = p2[0];
		newtet[5] = halfface[hf1[1]].pointhandle[0];
		newtet[6] = halfface[hf1[1]].pointhandle[1];
		newtet[7] = halfface[hf1[1]].pointhandle[2];

		for (int k = 0; k < 2; k ++)
		{
			for (int i = 0; i < 4; i++)
				for (int j = 0; j < 3; j++)
					p[i][j] = points[newtet[k*4+i]*3+j];
			qualnewtet[k] = tetquality(p, qualmeasure);
		}

		minqualafter1 = 1.0;
		for (int i = 0; i < 2; i ++)
		{
			if (minqualafter1 > qualtet1[i])
				minqualafter1 = qualtet1[i];
			if (minqualafter1 > qualnewtet[i])
				minqualafter1 = qualnewtet[i];
		}

		// ��ϲ��Զ���e2��Χ����������+�µ�����������
		newtet[8]  = p1[0];
		newtet[9]  = halfface[hf2[0]].pointhandle[0];
		newtet[10] = halfface[hf2[0]].pointhandle[1];
		newtet[11] = halfface[hf2[0]].pointhandle[2];

		newtet[12] = p1[0];
		newtet[13] = halfface[hf2[1]].pointhandle[0];
		newtet[14] = halfface[hf2[1]].pointhandle[1];
		newtet[15] = halfface[hf2[1]].pointhandle[2];

		for (int k = 2; k < 4; k ++)
		{
			for (int i = 0; i < 4; i++)
				for (int j = 0; j < 3; j++)
					p[i][j] = points[newtet[k*4+i]*3+j];
			qualnewtet[k] = tetquality(p, qualmeasure);
		}

		minqualafter2 = 1.0;
		for (int i = 0; i < 2; i ++)
		{
			if (minqualafter2 > qualtet2[i])
				minqualafter2 = qualtet2[i];
			if (minqualafter2 > qualnewtet[i])
				minqualafter2 = qualnewtet[i];
		}

		// update information
		if (minqualafter2 > minqualafter1 && minqualafter2 > minqualbefore)
		{
			tetra[tidx].strategy = 2;
			tetra[tidx].val = minqualafter2;
		}
		else if (minqualafter1 > minqualafter2 && minqualafter1 > minqualbefore)
		{
			tetra[tidx].strategy = 1;
			tetra[tidx].val = minqualafter1;
		}
		return true;
	}
	return false;
}

/* ȷ����������������Χ��������ֲ��Ƿ񸴺�Ҫ�� */
__device__ bool trianglesituationcheck(float *points, cu_tetra *tetra, int tidx, cu_flip23face *face, cu_halfface *halfface, int vidx, int qualmeasure)
{
	// ��vh_��Χ���������ڵ������������ཻ��һ��
	int hf[3];
	hf[0] = (tidx<<2)+(vidx+1)%4;
	hf[1] = (tidx<<2)+(vidx+2)%4;
	hf[2] = (tidx<<2)+(vidx+3)%4;

	int ophf[3];
	int ph[3];
	int tetidx[3];
	cu_flip23face tmpface;
	cu_tetra tet[3];
	cu_tetra currtet;
	for (int i = 0; i < 3; i ++)
	{
		// get opposite halfface
		tmpface = face[halfface[hf[i]].face];
		if (tmpface.hf[0] == hf[i])
			ophf[i] = tmpface.hf[1];
		else
			ophf[i] = tmpface.hf[0];

		// get the tetra contains the opposite halfface
		tet[i] = tetra[ophf[i]>>2];
		tetidx[i] = ophf[i]>>2;
		// get the top point
		ph[i] = tet[i].v[ophf[i]%4]; 
	}

	if (ph[0] == ph[1] && ph[0] == ph[2])
	{
		float p[4][3];
		float minqualbefore, minqualafter, tetqual;
		int newtet[4];
		cu_halfface tmphf;
		int hface;

		// calculate quality before newflip
		minqualbefore = 1.0;
		for (int k = 0; k < 3; k ++)
		{
			for (int i = 0; i < 4; i++)
				for (int j = 0; j < 3; j++)
					p[i][j] = points[tet[k].v[i]*3+j];
			tetqual = tetquality(p, qualmeasure);
			if (minqualbefore > tetqual)
				minqualbefore = tetqual;
		}

		currtet = tetra[tidx];
		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 3; j++)
				p[i][j] = points[currtet.v[i]*3+j];
		tetqual = tetquality(p, qualmeasure);
		if (minqualbefore > tetqual)
			minqualbefore = tetqual;

		tetra[tidx].quality = minqualbefore;
		tetra[tidx].fliptype = 2;
		tetra[tidx].newflipface[0] = (tidx<<2)+vidx;
		tetra[tidx].newflipface[1] = hf[0];
		tetra[tidx].newflipface[2] = hf[1];
		tetra[tidx].newflipface[3] = hf[2];
		tetra[tidx].tet[0] = tetidx[0];
		tetra[tidx].tet[1] = tetidx[1];
		tetra[tidx].tet[2] = tetidx[2];
		tetra[tidx].flippoint[0] = ph[0];

		// newtet
		hface = (tidx<<2)+vidx;
		tmphf = halfface[hface];
		newtet[0] = ph[0];
		newtet[1] = tmphf.pointhandle[0];
		newtet[2] = tmphf.pointhandle[1];
		newtet[3] = tmphf.pointhandle[2];

		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 3; j++)
				p[i][j] = points[newtet[i]*3+j];
		minqualafter = tetquality(p, qualmeasure);

		if (minqualafter > minqualbefore)
			tetra[tidx].val = minqualafter;
		return true;
	}
	return false;
}

__global__ void newflip_explore(float *points, int pointcnt, cu_tetra *tetra, int tetcnt, cu_flip23face *face, int facecnt, 
					            cu_halfface *halfface, int halffacecnt, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;

	int p[4];
	int trianglepoint[3];
	int pp;
	int flag;
	int state[3];         // ����λ��ͳ�� 0: outside   1: inside   2: on the boundary
	int vhandle;             // ����������Ķ���
	int situation;        // 0�������� 1��������  -1: ����
	int e1[2], e2[2];
	//bool issuitable;
	cu_tetra currtet;
	//int count = 0;

	while(tid < tetcnt)
	{
		currtet = tetra[tid];
		if (currtet.isboundary)
		{
			tid += offset;
			continue;
		}

		// get local data
		for (int i = 0; i < 4; i++)
			p[i] = currtet.v[i];

		situation = 0;
		state[0] = state[1] = state[2] = 0;

		/* ȷ����������״ */
		for (int i = 0; i < 4; i++)
		{
			// ��һ��ͶӰ���������������ڵ�ƽ��
			pp = p[i];

			for (int j = 0; j < 3; j++)
				trianglepoint[j] = p[(i+j+1)%4];

			vertexTriangleDistance(points, trianglepoint, pp, flag);

			if (flag == 0)
				++state[0];
			else if (flag == 1)
			{
				++state[1];
				vhandle = i;    // �����ż�¼
			}
			else
				++state[2];
		}

		if (state[1] == 0)
			situation = 0;
		else if (state[1] == 1)
			situation = 1;
		else
			situation = -1;

		// ȷ����������Χ������ֲ����
		if(situation == 0)//������
		{
// 			cu_tetra tet;
// 			tet = tetra[tid];

			// get tet points
			int v[4];
			//float p[4][3];
			//for (int i = 0; i < 4; i++)
			//{
			//	v[i] = tetra[tid].v[i];
			//	for (int j = 0; j < 3; j++)
			//		p[i][j] = points[v[i]*3+j];
			//}

			// get tet edges
			int edge[6][2];
			edge[0][0] = 0; edge[0][1] = 1;
			edge[1][0] = 0; edge[1][1] = 2;
			edge[2][0] = 0; edge[2][1] = 3;
			edge[3][0] = 2; edge[3][1] = 3;
			edge[4][0] = 1; edge[4][1] = 3;
			edge[5][0] = 1; edge[5][1] = 2;

			for (int i = 0; i < 3; i ++)
			{
				e1[0] = v[edge[i][0]];
				e1[1] = v[edge[i][1]];

				e2[0] = v[edge[(i+3)%6][0]];
				e2[1] = v[edge[(i+3)%6][1]];
				kitesituationcheck(points,tetra, tid, face, halfface, e1, e2, qualmeasure);
			}
		}
		else if(situation == 1)//������
		{
			trianglesituationcheck(points, tetra, tid, face, halfface, vhandle, qualmeasure);
		}
		tid += offset;
	}
}

void newfliptetraSelecting(struct cu_tetra *tetra, int tetcnt, int *selecttet, int &selecttetcnt)
{
	int stetcnt = 0;
	int* stet;
	int i, j;
	//float threashold = 1.0e-5;

	stet = new int[tetcnt];
	/* pick out the tets by flipping succeed val and sort it*/
	for (i = 0; i < tetcnt; i++)
	{
		if (tetra[i].val < 0 || !(tetra[i].val > tetra[i].quality))
			continue;

		// insert the tet
		for (j = stetcnt-1; j > -1 ; j--)
		{
			if (tetra[stet[j]].quality > tetra[i].quality)
				stet[j+1] = stet[j];
			else 
				break;
		}
		stet[j+1] = i;
		++ stetcnt;
	}

	/* if there are no any face meeting the requirement */
	if (!stetcnt)
	{
		selecttetcnt = 0;
		selecttet = NULL;
		return;
	}

	/* select a new tet set */
	int tet[4];
	bool *tetflag;
	tetflag = new bool[tetcnt];
	memset(tetflag, 1, tetcnt*sizeof(bool));

	// push the first tet
	selecttet[0] = stet[0];
	selecttetcnt = 1;

	// set flags of tets incident to the tet
	tetflag[selecttet[0]] = 0;
	if (tetra[selecttet[0]].fliptype == 1)
	{
		for (i = 0; i < 4; i ++)
			tetflag[tetra[selecttet[0]].tet[i]] = 0;
	}
	else
	{
		for (i = 0; i < 3; i ++)
			tetflag[tetra[selecttet[0]].tet[i]] = 0;
	}

	/* if the tets incident to a face are available, 
	   then add the face into the array*/
	for (i = 1; i < stetcnt; i++)
	{
		if (tetra[stet[i]].fliptype == 1)
		{
			for (j = 0; j < 4; j ++)
				tet[j] = tetra[stet[i]].tet[j];
			if (tetflag[tet[0]] && tetflag[tet[1]] && tetflag[tet[2]] && tetflag[tet[3]] && tetflag[stet[i]])
			{
				selecttet[selecttetcnt++] = stet[i];
				tetflag[stet[i]] = 0;
				tetflag[tet[0]] = 0;
				tetflag[tet[1]] = 0;
				tetflag[tet[2]] = 0;
				tetflag[tet[3]] = 0;
			}
		}
		else
		{
			for (j = 0; j < 3; j ++)
				tet[j] = tetra[stet[i]].tet[j];
			if (tetflag[tet[0]] && tetflag[tet[1]] && tetflag[tet[2]] && tetflag[stet[i]])
			{
				selecttet[selecttetcnt++] = stet[i];
				tetflag[stet[i]] = 0;
				tetflag[tet[0]] = 0;
				tetflag[tet[1]] = 0;
				tetflag[tet[2]] = 0;
			}
		}
	}
}

__global__ void newflip(cu_tetra *tetra, int *selecttet, int selecttetcnt, cu_halfface *halfface)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;

	int fliptype;
	cu_tetra currtet;
	//float p[4][3];
	//float qual;
	//float newqual[2];
	while(tid < selecttetcnt)
	{
		currtet = tetra[selecttet[tid]];
		fliptype = currtet.fliptype;

		// ������
		if (fliptype == 1)
		{
			int hf1[2], hf2[2];
			int tet1[2], tet2[2];
			int p1, p2;
			int newtet[8];

			hf1[0] = currtet.newflipface[0];
			hf1[1] = currtet.newflipface[1];
			hf2[0] = currtet.newflipface[2];
			hf2[1] = currtet.newflipface[3];
			tet1[0] = currtet.tet[0];
			tet1[1] = currtet.tet[1];
			tet2[0] = currtet.tet[2];
			tet2[1] = currtet.tet[3];
			p1 = currtet.flippoint[0];
			p2 = currtet.flippoint[1];

			if (currtet.strategy == 1)
			{
				newtet[0] = p2;
				newtet[1] = halfface[hf1[0]].pointhandle[0];
				newtet[2] = halfface[hf1[0]].pointhandle[1];
				newtet[3] = halfface[hf1[0]].pointhandle[2];

				newtet[4] = p2;
				newtet[5] = halfface[hf1[1]].pointhandle[0];
				newtet[6] = halfface[hf1[1]].pointhandle[1];
				newtet[7] = halfface[hf1[1]].pointhandle[2];

				for (int i = 0; i < 2; i ++)
				{
					for (int j = 0; j < 4; j++)
						tetra[tet2[i]].v[j] = newtet[i*4+j];
				}
			}
			else
			{
				newtet[0] = p1;
				newtet[1] = halfface[hf2[0]].pointhandle[0];
				newtet[2] = halfface[hf2[0]].pointhandle[1];
				newtet[3] = halfface[hf2[0]].pointhandle[2];

				newtet[4] = p1;
				newtet[5] = halfface[hf2[1]].pointhandle[0];
				newtet[6] = halfface[hf2[1]].pointhandle[1];
				newtet[7] = halfface[hf2[1]].pointhandle[2];

				for (int i = 0; i < 2; i ++)
				{
					for (int j = 0; j < 4; j++)
						tetra[tet1[i]].v[j] = newtet[i*4+j];
				}
			}
		}
		// ������
		else
		{
			int ph;
			int hf;
			int tetidx[3];
			int newtet[4];

			ph = currtet.flippoint[0];
			hf = currtet.newflipface[0];
			for (int i = 0; i < 3; i++)
				tetidx[i] = currtet.tet[i];

			// newtet
			newtet[0] = ph;
			newtet[1] = halfface[hf].pointhandle[0];
			newtet[2] = halfface[hf].pointhandle[1];
			newtet[3] = halfface[hf].pointhandle[2];

			for (int i = 0; i < 4; i++)
			{
				tetra[tetidx[0]].v[i] = newtet[i];
				tetra[tetidx[1]].v[i] = -1;
				tetra[tetidx[2]].v[i] = -1;
			}
		}
		for (int i = 0; i < 4; i++)
			tetra[tid].v[i] = -1;

		tid += offset;
	}
}

extern "C" void cuda_newflip(float *points, int pointcnt, int *meshtets, int tetcnt, int *face, int facecnt, 
							 int *halfface, int halffacecnt, int qualmeasure, float& qualbefore_, float& qualafter_, 
							 int &flipsucc, float &time)
{
	// create face and halfface structure
	struct cu_flip23face *cface;
	struct cu_halfface *chalfface;
	struct cu_tetra *ctetra;
	float qualbefore;
	float qualafter;
	//float qualtmp;
	int tetcapacity;
	int facecapacity;
	int halffacecapacity;

	tetcapacity = int(tetcnt);
	facecapacity = int(facecnt);
	halffacecapacity = int(halffacecnt);
	ctetra = new struct cu_tetra[tetcapacity];
	cface = new struct cu_flip23face[facecapacity];
	chalfface = new struct cu_halfface[halffacecapacity];

	// get face
	for (int i = 0; i < facecnt; i++)
	{
		cface[i].hf[0] = face[2*i];
		cface[i].hf[1] = face[2*i+1];
		cface[i].quality = -1;
		cface[i].val = -1;
	}

	// get halfface
	for (int i = 0; i < halffacecnt; i++)
	{
		chalfface[i].pointhandle[0] = halfface[4*i];
		chalfface[i].pointhandle[1] = halfface[4*i+1];
		chalfface[i].pointhandle[2] = halfface[4*i+2];
		chalfface[i].face = halfface[4*i+3];
	}

	// get tetra
	for (int i = 0; i < tetcnt; i ++)
	{
		ctetra[i].v[0] = meshtets[5*i];
		ctetra[i].v[1] = meshtets[5*i+1];
		ctetra[i].v[2] = meshtets[5*i+2];
		ctetra[i].v[3] = meshtets[5*i+3];
		ctetra[i].isboundary = meshtets[5*i+4];
		ctetra[i].fliptype = -1;
		ctetra[i].newflipvertex = -1;
		ctetra[i].quality = -1;
		ctetra[i].val = -1;
		for (int j = 0; j < 4; j ++)
			ctetra[i].newflipface[j] = 0;
	}

	// calculate quality before flipping
	cutetramesh_quality(points, pointcnt, ctetra, tetcnt, qualmeasure, qualbefore);

	// CUDA Parallel
	// �����豸�洢�ռ�
	//int loop = 1;
	float *dev_points;
	//int *dev_meshtets, *dev_tempmeshtets;
	struct cu_flip23face *dev_face;
	struct cu_halfface *dev_halfface;
	struct cu_tetra *dev_tetra;
	int *selecttetra;
	int selecttetracnt;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// flip23 �������µ������塢������棬�����Ԥ����һ���ֿռ�
	hipMalloc((void**)&dev_points, 3*pointcnt*sizeof(float));
	//hipMalloc((void**)&dev_meshtets, 4*tetcapacity*sizeof(int));
	hipMalloc((void**)&dev_face, facecapacity*sizeof(cu_flip23face));
	hipMalloc((void**)&dev_halfface, halffacecapacity*sizeof(cu_halfface));
	hipMalloc((void**)&dev_tetra, tetcapacity*sizeof(cu_tetra));

	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(dev_meshtets, meshtets, 4*tetcnt*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_face, cface, facecnt*sizeof(cu_flip23face), hipMemcpyHostToDevice);
	hipMemcpy(dev_halfface, chalfface, halffacecnt*sizeof(cu_halfface), hipMemcpyHostToDevice);
	hipMemcpy(dev_tetra, ctetra, tetcnt*sizeof(cu_tetra), hipMemcpyHostToDevice);

	// do new flip
	// newflip explore
	int blocks = imin(tetcnt, BlockPerGrid);
	newflip_explore<<<blocks, (tetcnt+blocks-1)/blocks>>>(dev_points, pointcnt, dev_tetra, tetcnt, dev_face, facecnt, 
		dev_halfface, halffacecnt, qualmeasure); 
	hipMemcpy(ctetra, dev_tetra, tetcnt*sizeof(cu_tetra), hipMemcpyDeviceToHost);

	// tetra selecting
	selecttetra = new int[tetcnt];
	selecttetracnt = 0;
	newfliptetraSelecting(ctetra, tetcnt, selecttetra, selecttetracnt);

	// newflip (set -1 to the meshtets[] of invalid tets)
	if (selecttetracnt)
	{
		int *dev_selecttetra;
		hipMalloc((void**)&dev_selecttetra, selecttetracnt*sizeof(int));
		hipMemcpy(dev_selecttetra, selecttetra, selecttetracnt*sizeof(int), hipMemcpyHostToDevice);

		// do newflip
		blocks = imin(selecttetracnt, BlockPerGrid);
		newflip<<<blocks, (selecttetracnt+blocks-1)/blocks>>>(dev_tetra, dev_selecttetra, selecttetracnt, dev_halfface);

		flipsucc += selecttetracnt;
		hipFree(dev_selecttetra);
	}		

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elaspsedTime;
	hipEventElapsedTime(&elaspsedTime, start, stop);

	time = elaspsedTime;

	// calculate quality after flipping
	qualafter = 1.0;
	hipMemcpy(ctetra, dev_tetra, 4*tetcnt*sizeof(int), hipMemcpyDeviceToHost);
	// calculate quality before flipping
	cutetramesh_quality(points, pointcnt, ctetra, tetcnt, qualmeasure, qualafter);

	qualbefore_ = qualbefore;
	qualafter_ = qualafter;
	flipsucc = selecttetracnt;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_points);
	//hipFree(dev_meshtets);
	hipFree(dev_halfface);
	hipFree(dev_face);

	delete [] cface;
	delete [] chalfface;
	delete [] ctetra;
	delete [] selecttetra;
}
/************************ End of New Flip *****************************/

/************************ Edge Contraction ****************************/

__device__ void getECNewTetras(int *newtetra, int *newtetracnt, int *fromtetra, int fromtetracnt, int *totetra, int totetracnt,
							   int *edgestar, int edgestarcnt)
{
	*newtetracnt = 0;
	int i, j, k;
	int currft, currtt;
	for (i = 0, j = 0; i < fromtetracnt || j < totetracnt; i++, j++)
	{
		if (i < fromtetracnt)
			currft = fromtetra[i];
		else
			currft = -1;

		if (j < totetracnt)
			currtt = totetra[j];
		else
			currtt = -1;

		for (k = 0; k < edgestarcnt; k++)
		{
			if (edgestar[k] == fromtetra[i])
				currft = -1;
			if(edgestar[k] == totetra[j])
				currtt = -1;
		}

		if (currft != -1)
			newtetra[(*newtetracnt)++] = currft;
		if (currtt != -1)
			newtetra[(*newtetracnt)++] = currtt;
	}
}

// for model: out, rand1, rand2
//#define MAXPOINTSTAR 300
//#define MAXPOINTSTARSUM 550

// for other models
#define MAXPOINTSTAR 70
#define MAXPOINTSTARSUM 120

#define MASEDGESTAR 25

// not tested yet
__global__ void edge_contraction_explore(float *points, int *meshtets, cu_edge *edge, int edgecnt, cu_halfedge *halfedge, 
										 int *incidenttet, int *incidenttetcnt, int largesttetcnt, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int fromv, tov, tempv;
	float fp[3], tp[3], mp[3];
	int edgestar[MASEDGESTAR];
	int newtetra[MAXPOINTSTARSUM];
	int fromtetra[MAXPOINTSTAR];
	int totetra[MAXPOINTSTAR];
	int edgestarcnt;
	int newtetracnt;
	int fromtetracnt;
	int totetracnt;
	float newtetrapoint[MAXPOINTSTARSUM*12];
	int *phe;
	float tempqual;
	float qualbefore, qualafter;
	cu_edge curredge;
	int i, j, k;

	while(tid < edgecnt)
	{
		curredge = edge[tid];

		// if the edge is boundary go next
		if (curredge.is_boundary) 
		{
			tid += offset;
			continue;
		}

		/* fetch local data */
		// get edge star
		phe = curredge.halfedge;
		edgestarcnt = curredge.halfedgecnt / 2;
		for (i = 0; i < edgestarcnt; i ++)
			edgestar[i] = (phe[i*2])/12;

		// get new tetras
		fromv = halfedge[phe[0]].fromv;
		tov = halfedge[phe[0]].tov;
		fromtetracnt = incidenttetcnt[fromv];
		totetracnt = incidenttetcnt[tov];

		for (i = 0; i < fromtetracnt; i++)
			fromtetra[i] = incidenttet[fromv*largesttetcnt + i];
		for (i = 0; i < totetracnt; i++)
			totetra[i] = incidenttet[tov*largesttetcnt + i];

		getECNewTetras(newtetra, &newtetracnt, fromtetra, fromtetracnt, totetra, totetracnt, edgestar, edgestarcnt);

		// calculate quality
		qualbefore = minstackquality(points, meshtets, newtetra, newtetracnt, qualmeasure);
		tempqual = minstackquality(points, meshtets, edgestar, edgestarcnt, qualmeasure);
		qualbefore = qualbefore < tempqual ? qualbefore : tempqual;

		// get endpoints
		for (i = 0; i < 3; i ++)
		{
			fp[i] = points[3*fromv+i];
			tp[i] = points[3*tov + i];
			mp[i] = (fp[i] + tp[i])/2.0;
		}

		// get newtetras' points
		for (i = 0; i < newtetracnt; i++)
		{
			for (j = 0; j < 4; j++)
			{
				tempv = meshtets[4*newtetra[i]+j];
				if (tempv == fromv || tempv == tov)
				{
					for (k = 0; k < 3; k++)
						newtetrapoint[3*(i*4+j)+k] = mp[k];
				}
				else
				{
					for (k = 0; k < 3; k++)
						newtetrapoint[3*(i*4+j)+k] = points[3*tempv+k];
				}
			}
		}

		// calculate new quality
		qualafter = minstackquality(newtetrapoint, newtetracnt, qualmeasure);

		// record data
		edge[tid].quality = qualbefore;
		edge[tid].val = qualafter - qualbefore;

		tid += offset;
	}
}


__global__ void edge_contraction(float *points, int *meshtets, cu_edge *edge, cu_halfedge *halfedge,
								 int *selectedge, int selectedgecnt)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;

	int fromv, tov;
	float fp[3], tp[3], mp[3];
	cu_edge curredge;
	int tetidx;
	int i, j;

	while(tid < selectedgecnt)
	{
		curredge = edge[selectedge[tid]];

		// set two endpoints with new coordinates
		fromv = halfedge[curredge.halfedge[0]].fromv;
		tov = halfedge[curredge.halfedge[0]].tov;
		for (i = 0; i < 3; i++)
		{
			fp[i] = points[3*fromv+i];
			tp[i] = points[3*tov+i];
			mp[i] = (fp[i] + tp[i])/2.0;
			points[3*fromv+i] = mp[i];
			points[3*tov+i] = mp[i];
		}

		// set invalid data to tetras incident to current edge
		for (i = 0; i < curredge.halfedgecnt; i++)
		{
			tetidx = (curredge.halfedge[i])/4;
			for (j = 0; j < 4; j++)
			{
				meshtets[4*tetidx+j] = -1;
			}
		}
		tid += offset;
	}
}

void edgeSelecting_EC(cu_edge *cuedge, int edgecnt, int tetcnt, int *selectedge, int &selectedgecnt)
{
	int sedgecnt = 0;
	int* sedge = new int[edgecnt];
	int i, j;
	float threashold = 1.0e-5;
	bool isavailable;
	cu_edge curredge;

	/* pick out the edges by flipping succeed val
	and sort it*/
	for (i = 0; i < edgecnt; i++)
	{
		if (!(cuedge[i].val > threashold))
			continue;

		// insert the edge
		for (j = sedgecnt-1; j > -1 ; j--)
		{
			if (cuedge[sedge[j]].quality > cuedge[i].quality)
				sedge[j+1] = sedge[j];
			else 
				break;
		}
		sedge[j+1] = i;
		++ sedgecnt;
	}

	/* if there are no any edge meeting the requirement */
	if (!sedgecnt)
	{
		selectedgecnt = 0;
		selectedge = NULL;
		return;
	}

	/* select a new edge set in which any two of them not in a same tet */
	//int tet;
	bool *tetflag;
	tetflag = new bool[tetcnt];
	memset(tetflag, true, tetcnt*sizeof(bool));

	// push the first edge
	selectedge[0] = sedge[0];
	selectedgecnt = 1;

	// set flags of tets incident to the first edge
	curredge = cuedge[sedge[0]];
	for (i = 0; i < curredge.halfedgecnt; i++)
	{
		tetflag[(curredge.halfedge[i])/12] = false;
	}

	/* if the tets incident to a edge are available, 
	then add the edge into the array*/
	for (i = 1; i < sedgecnt; i++)
	{
		isavailable = true;
		curredge = cuedge[sedge[i]];
		for (j = 0; j < curredge.halfedgecnt; j++)
		{
			if (!tetflag[(curredge.halfedge[j])/12])
			{
				isavailable = false;
				break;
			}
		}
		if (isavailable)
		{
			selectedge[selectedgecnt++] = sedge[i];
			for (j = 0; j < curredge.halfedgecnt; j++)
			{
				tetflag[(curredge.halfedge[j])/12] = false;
			}
		}
	}
	delete [] tetflag;
}

void edgeSelecting_EC_1(cu_edge *cuedge, int edgecnt, cu_halfedge *cuhalfedge, int halfedgecnt, int tetcnt, 
					  int *incidenttet, int *incidenttetcnt, int largesttetcnt, int *selectedge, int &selectedgecnt)
{
	int sedgecnt = 0;
	int* sedge = new int[edgecnt];
	int i, j;
	int fromv, tov;
	float threashold = 1.0e-5;
	bool isavailable;
	cu_edge curredge;

	/* pick out the edges by flipping succeed val
	and sort it*/
	for (i = 0; i < edgecnt; i++)
	{
		if (!(cuedge[i].val > threashold))
			continue;

		// insert the edge
		for (j = sedgecnt-1; j > -1 ; j--)
		{
			if (cuedge[sedge[j]].quality > cuedge[i].quality)
				sedge[j+1] = sedge[j];
			else 
				break;
		}
		sedge[j+1] = i;
		++ sedgecnt;
	}

	/* if there are no any edge meeting the requirement */
	if (!sedgecnt)
	{
		selectedgecnt = 0;
		selectedge = NULL;
		return;
	}

	/* select a new edge set in which any two of them not in a same tet */
	//int tet;
	bool *tetflag;
	tetflag = new bool[tetcnt];
	memset(tetflag, true, tetcnt*sizeof(bool));

	// push the first edge
	selectedge[0] = sedge[0];
	selectedgecnt = 1;

	// set flags of tets incident to the first edge
	curredge = cuedge[sedge[0]];
	fromv = cuhalfedge[curredge.halfedge[0]].fromv;
	tov = cuhalfedge[curredge.halfedge[0]].tov;
	for (i = 0; i < incidenttetcnt[fromv]; i++)
	{
		tetflag[incidenttet[fromv*largesttetcnt+i]] = false;
	}
	for (i = 0; i < incidenttetcnt[tov]; i++)
	{
		tetflag[incidenttet[tov*largesttetcnt+i]] = false;
	}

	/* if the tets incident to a edge are available, 
	then add the edge into the array*/
	for (i = 1; i < sedgecnt; i++)
	{
		isavailable = true;
		curredge = cuedge[sedge[i]];
		fromv = cuhalfedge[curredge.halfedge[0]].fromv;
		tov = cuhalfedge[curredge.halfedge[0]].tov;
		for (j = 0; j < incidenttetcnt[fromv] && isavailable; j++)
		{
			if (!tetflag[incidenttet[fromv*largesttetcnt+j]])
				isavailable = false;
		}
		for (j = 0; j < incidenttetcnt[tov] && isavailable; j++)
		{
			if (!tetflag[incidenttet[tov*largesttetcnt+j]])
				isavailable = false;
		}

		if (isavailable)
		{
			selectedge[selectedgecnt++] = sedge[i];
			for (j = 0; j < incidenttetcnt[fromv]; j++)
			{
				tetflag[incidenttet[fromv*largesttetcnt+j]] = false;
			}
			for (j = 0; j < incidenttetcnt[tov]; j++)
			{
				tetflag[incidenttet[tov*largesttetcnt+j]] = false;
			}
		}
	}
	delete [] tetflag;
}


extern "C" void cuda_edgeContraction(float *points, int pointcnt, int *meshtets, int tetcnt, int *edge, int edgecnt,
									 int *halfedge, int halfedgecnt, int *incidenttet, int* incidenttetcnt, int &largesttetcnt, 
									 int qualmeasure, float &qualbefore_, float &qualafter_, int &succ_, float &time)
{
	//FILE *outfile = NULL;
	//outfile = fopen("F:\\kernel_data_output.txt", "w");

	cu_edge *cedge;
	cu_halfedge *chalfedge;
	cedge = new cu_edge[edgecnt];
	chalfedge = new cu_halfedge[halfedgecnt];


	float qualbefore, qualafter;
	float *dev_points;
	int *dev_meshtets;
	int *dev_incidenttet;
	int *dev_incidenttetcnt;
	struct cu_edge *dev_edge;
	struct cu_halfedge *dev_halfedge;
	int *selectedge;
	int selectedgecnt;
	int loop = 1;

	// calculate quality before edge contraction
	cuda_tetquality(points, pointcnt, meshtets, tetcnt, qualmeasure, qualbefore);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// build edge
	int idx = 0;
	for (int i = 0; i < edgecnt; i ++)
	{
		cedge[i].is_boundary = edge[idx++];
		cedge[i].halfedgecnt = edge[idx++];
		for (int j = 0; j < cedge[i].halfedgecnt; j++)
		{
			cedge[i].halfedge[j] = edge[idx++];
		}
		cedge[i].quality = cedge[i].val = -1;
	}

	// build halfedge
	for (int i = 0; i < halfedgecnt; i++)
	{
		chalfedge[i].edgehandle = halfedge[i*3];
		chalfedge[i].fromv = halfedge[i*3+1];
		chalfedge[i].tov = halfedge[i*3+2];
	}

 	hipMalloc((void**)&dev_points, 3*pointcnt*sizeof(float));
 	hipMalloc((void**)&dev_meshtets, 4*tetcnt*sizeof(int));
 	hipMalloc((void**)&dev_edge, edgecnt*sizeof(cu_edge));
 	hipMalloc((void**)&dev_halfedge, halfedgecnt*sizeof(cu_halfedge));
 	hipMalloc((void**)&dev_incidenttet, pointcnt * largesttetcnt * sizeof(int));
 	hipMalloc((void**)&dev_incidenttetcnt, pointcnt * sizeof(int));
 
 	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
 	hipMemcpy(dev_meshtets, meshtets, 4*tetcnt*sizeof(int), hipMemcpyHostToDevice);
 	hipMemcpy(dev_edge, cedge, edgecnt*sizeof(cu_edge), hipMemcpyHostToDevice);
 	hipMemcpy(dev_halfedge, chalfedge, halfedgecnt*sizeof(cu_halfedge), hipMemcpyHostToDevice);
 	hipMemcpy(dev_incidenttet, incidenttet, pointcnt * largesttetcnt * sizeof(int), hipMemcpyHostToDevice);
 	hipMemcpy(dev_incidenttetcnt, incidenttetcnt, pointcnt * sizeof(int), hipMemcpyHostToDevice);

	int succ = 0;

	//InitGPUSet();
	//cuPrintInit();
	while(loop)
	{
		/***********do parallel edge contraction**********/
		// edge contraction explore
		int blocks = imin(edgecnt, BlockPerGrid);
		edge_contraction_explore<<<blocks, (edgecnt+blocks-1)/blocks>>>(dev_points, dev_meshtets, dev_edge, edgecnt, dev_halfedge, dev_incidenttet, dev_incidenttetcnt, largesttetcnt, qualmeasure);
		//cudaPrintfDisplay(outfile, true);
		//cudaPrintfEnd(); 
		//fclose(outfile);

		hipMemcpy(cedge, dev_edge, edgecnt*sizeof(cu_edge), hipMemcpyDeviceToHost);

		// edge selecting
		selectedge = new int[edgecnt];
		edgeSelecting_EC(cedge, edgecnt, tetcnt, selectedge, selectedgecnt);
		//edgeSelecting_EC_1(cedge, edgecnt, chalfedge, halfedgecnt, tetcnt, incidenttet, incidenttetcnt, largesttetcnt, selectedge, selectedgecnt);

		// edge contraction (set -1 to the meshtets[] of invalid tets)
		if (selectedgecnt)
		{
			int *dev_selectedge;
			hipMalloc((void**)&dev_selectedge, selectedgecnt*sizeof(int));
			hipMemcpy(dev_selectedge, selectedge, selectedgecnt*sizeof(int), hipMemcpyHostToDevice);

			// do edge contraction
			blocks = imin(selectedgecnt, BlockPerGrid);
			edge_contraction<<<blocks, (selectedgecnt+blocks-1)/blocks>>>(dev_points, dev_meshtets, dev_edge, dev_halfedge, dev_selectedge, selectedgecnt);

			succ += selectedgecnt;
			hipFree(dev_selectedge);
		}
		-- loop;
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elaspsedTime;
	hipEventElapsedTime(&elaspsedTime, start, stop);

	time = elaspsedTime;

	// calculate quality after flipping
	qualafter = 1.0;
	hipMemcpy(meshtets, dev_meshtets, 4*tetcnt*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(points, dev_points, 3*pointcnt*sizeof(float), hipMemcpyDeviceToHost);
	cuda_tetquality(points, pointcnt, meshtets, tetcnt, qualmeasure, qualafter);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_points);
	hipFree(dev_meshtets);
	hipFree(dev_halfedge);
	hipFree(dev_edge);
	hipFree(dev_incidenttet);
	hipFree(dev_incidenttetcnt);

	delete [] cedge;
	delete [] chalfedge;

	qualbefore_ = qualbefore;
	qualafter_ = qualafter;
	succ_ = succ;
}
/************************ End of Edge Contraction ************************/

/************************ Parallel Vertex Insertion ***************************/

/* find tet adjacencies to a specified face 
* param[hf]: handle of the specified half face
* param[outin]: return the two adjacencies tets 
outin[0]: tet contains the hf's opposite face
outin[1]: tet contains the face hf
*/
__device__ int opposite_halfface(cu_face *face, cu_halfface *halfface, int hf)
{
	int fidx = halfface[hf].face;
	int oppf;
	if (face[fidx].hf[0] == hf)
		oppf = face[fidx].hf[1];
	else
		oppf = face[fidx].hf[0];
	return oppf;
}


/* find cavitytet by a tetrahedron handle*/
__device__ int findCavityTet(CavityTet *outcavity, int outcavitysize, int tethandle)
{
	int idx;
	for (idx = 0; idx < outcavitysize; idx++)
	{
		if (outcavity[idx].handle == tethandle) break;
	}
	return idx;
}

__device__ int find_array_int(int *array_, int arrsize, int data_)
{
	int i = 0;
	for (; i < arrsize; i++)
	{
		if (array_[i] == data_)
			break;
	}
	return i;
}

__device__ void halfFacePoints(float *point, cu_halfface *halfface, int hf, float fpoint[3][3])
{
	for (int i = 0; i < 3; i++)
	{
		fpoint[i][0] = point[(halfface[hf].pointhandle[i])*3];
		fpoint[i][1] = point[(halfface[hf].pointhandle[i])*3+1];
		fpoint[i][2] = point[(halfface[hf].pointhandle[i])*3+2];
	}
}

/* check if a facet (indicated by fp1, fp2, fp3) is oriented to a vertex v */
__device__ int orient(float v[3], float fp1[3], float fp2[3], float fp3[3])
{
	float center[3];
	float facenormal[3];
	float result;
	float vec1[3], vec2[3];
	
	// get face center
	vector_add(fp1, fp2, center);
	vector_add(center, fp3, center);
	center[0] /= 3.0;
	center[1] /= 3.0;
	center[2] /= 3.0;

	vector_minus(fp2, fp1, vec1);
	vector_minus(fp3, fp1, vec2);
	vector_cross(vec1, vec2, facenormal);

	vector_minus(v, center, vec1);
	result = vector_dot(vec1, facenormal);
	if (result > 0)
		return 1;
	if (result == 0)
		return 0;
	return -1;
}

/* given a vertex, it's position, an initial cavitiy of tets and 
* a set of outward-oriented faces of that cavity, build a DAG
* representing the largest star-shaped cavity from the point of
* view of the inserted vertex */
__device__ void buildcavitydag(float *point, cu_InsertTet *tets, cu_face *face, cu_halfface *halfface, 
					           float pnew[3], int inserttetidx, cu_InsertTet &inserttet, float tetpoint[4][3], int qualmeasure, 
				               CavityTet *outcavity, int &outcavitysize)
{
	int F[MAXSTACKFACE];    /* candidate face list */
	int W[MAXSTACKFACE];    /* wall face list */
	int B[MAXSTACKTET];     /* blocking tet list */
	int Fsize=0, Wsize=0, Bsize=0;
	int currtet;                   /* current tet */
	int currf;                     /* current face */
	int oppf;
	int outin[2];
	int Wcount = 0;
	bool facing = true;
	int otherfaces[3];
	int deepest = 0;
	int nonwall[3];
	int numnonwall = 0;
	float fpoint[3][3];

	/* output cavity stuff */
	CavityTet cavtet;
	CavityFace cavface;
	CavityFace cavface2;
	int tetidx;
	int i,j;

	/* initialize cavity tet*/
	cavtet.handle = inserttetidx;
	cavtet.quality = tets[inserttetidx].quality;
	cavtet.depth = 0;
	cavtet.outfacesize = 0;
	cavtet.parents[0] = NOCAVITYTET;
	cavtet.parents[1] = NOCAVITYTET;
	cavtet.parents[2] = NOCAVITYTET;
	outcavity[outcavitysize++] = cavtet;

	/* initialize candidate face list */
	for (i = 0; i < 4; i++)
	{
		F[Fsize++] = inserttetidx*4+i;
	}

	/* now, as long as we have candidate faces */
	while (Fsize > 0)
	{
		//cuPrintf("\nFaces in F set: \n");
		//for (i = 0; i < Fsize; i++)
		//{
		//	cuPrintf("%d ", F[i]);
		//}
		//cuPrintf("\n");

		if (outcavitysize > MAXCAVITYTETS || Fsize > MAXCAVITYFACES || 
			Wsize > MAXCAVITYFACES || Bsize > MAXCAVITYTETS)
			return;

		/* pull a face out of F */
		currf = F[Fsize-1];
		--Fsize;

		/* get t, the tet on the other side of this face 
		* outin[0]: outward facing tet; outin[1]: inward facing tet*/
		oppf = opposite_halfface(face, halfface, currf);
		if (oppf != NOCAVITYFACE)
			outin[0] = oppf/4;
		else
			outin[0] = NOCAVITYTET;
		outin[1] = currf/4;

		/* the inward facing tet should already be in the output cavity.
		* find it, and add this face as an outgoing face */
		tetidx = findCavityTet(outcavity, outcavitysize, outin[1]);
		if (tetidx == outcavitysize)
			return;

		/* compute the quality of the cavity tet with this face */
		halfFacePoints(point, halfface, currf, fpoint);
		
		cavface.handle = currf;
		cavface.quality = tetquality(pnew, fpoint[0], fpoint[1], fpoint[2], qualmeasure);

		/* check to make sure it's not a ghost tet */
		if (outin[0] == GHOSTTET)
		{
			/* note that this face has no child, and assign it to its parent tet */
			cavface.child = NOCAVITYTET;
			if (outcavity[tetidx].outfacesize < 5)
				outcavity[tetidx].outfaces[(outcavity[tetidx].outfacesize)++] = cavface;

			/* add this face to the wall list */
			W[Wsize++] = currf;
			continue;
		}

		// fetch the outward facing tet
		currtet = outin[0];

		// fetch the other faces of current tet
		int idx = 0;
		j = currtet*4+4;
		for (i = currtet*4; i < j; ++i)
		{
			/* except the opposite half face of the current face*/
			if (i == oppf)
				continue;
			if (idx < 3)
				otherfaces[idx ++] = i;
		}

		/* is t a cavity tet? */
		if (findCavityTet(outcavity, outcavitysize, currtet) != outcavitysize)
		{
			/* we need to add this face to the parent tet, indicating 
			that it has no child because the tet on the other side
			doesn't depend in it's removal to exist */
			cavface.child = NOCAVITYTET;
			if (outcavity[tetidx].outfacesize < 5)
				outcavity[tetidx].outfaces[(outcavity[tetidx].outfacesize)++] = cavface;

			/* yes, do nothing */
			continue;
		}

		/* is t a blocking tet? */
		if (find_array_int(B, Bsize, currtet) != Bsize)
		{
			/* if there is one other wall face of this tet, and the other two faces
			are visible from v, we can add this tet to the cavity. */
			Wcount = 1;
			facing = true;

			j = currtet*4+4;
			for (i = currtet*4; i < j; ++i)
			{
				/* except the current face */
				if (i == oppf)
					continue;

				/* is this face already marked as a wall ? 
				* testing the opposite face which may be visited before */
				if (find_array_int(W, Wsize, opposite_halfface(face, halfface, i)) != Wsize)
				{
					Wcount++;
				}
				else
					/* it's not a wall... is it oriented toward v? */
				{
					halfFacePoints(point, halfface, i, fpoint);
					if (orient(pnew, fpoint[0], fpoint[1], fpoint[2]) <= MINFACING)
					{
						facing = false;
					}
				}
			}

			/* only allow tets with three parents if we are allowing vertex deletion */
			if ((Wcount == 2 || Wcount == 3) && facing)
			{
				/* this tet can be added to the cavity */

				/* remove it from B */
				i = find_array_int(B, Bsize, currtet);
				if (i < Bsize)
				{
					B[i] = B[Bsize-1];
					-- Bsize;
				}
				/* add it to C */
				//C[Csize++] = currtet;

				/* add this tet to the output cavity */
				cavtet.handle = currtet;
				cavtet.outfacesize = 0;
				cavtet.quality = tets[currtet].quality;

				/* we know one parent must be the one we found above */
				cavtet.parents[0] = outcavity[tetidx].handle;
				/* the depth is one more than the parent depth */
				cavtet.depth = outcavity[tetidx].depth + 1;
				/* if this is a new deepest, remember it */
				if (cavtet.depth > deepest) deepest = cavtet.depth;
				outcavity[outcavitysize++] = cavtet;

				/* add this face to the parent tet with the correct child */
				//cavface.child = cavtet;
				if (outcavity[tetidx].outfacesize < 5)
					outcavity[tetidx].outfaces[outcavity[tetidx].outfacesize++] = cavface;

				/* remove any faces that were in W, add others to F. Handle output
				tet faces that need to be added */
				numnonwall = 0;

				/* first, handle all wall face so we can set the correct depth */
				j = currtet*4+4;
				for (i = currtet*4; i<j; ++ i)
				{
					/* except the current face */
					if (i == oppf)
						continue;

					int opphf = opposite_halfface(face, halfface, i);
					/* is this already a wall face? */
					int idx = find_array_int(W, Wsize, opphf);
					if (idx != Wsize)
					{
						W[idx] = W[Wsize-1];
						--Wsize;

						/* because this face was previously a wall face,
						it has some cavity tet that it belongs to. find
						this tet in the output cavity and set it's child face */
						idx = findCavityTet(outcavity, outcavitysize, opphf/4);
						if (idx == outcavitysize)
							return;

						/* add this face to the parent tet's outgoing faces */
						cavface.handle = opphf;
						halfFacePoints(point, halfface, opphf, fpoint);
						cavface.quality = tetquality(pnew, fpoint[0], fpoint[2], fpoint[1], qualmeasure);
						cavface.child = currtet;

						/* make sure that this face is already in this tet */
						if (outcavity[idx].outfacesize < 5)
							outcavity[idx].outfaces[outcavity[idx].outfacesize++] = cavface;

						/* assign the parent tet as the second parent of the new cavity tet */
						if (outcavity[outcavitysize-1].parents[1] == NOCAVITYTET)
						{
							outcavity[outcavitysize-1].parents[1] = outcavity[idx].handle;
						}
						else
						{
							outcavity[outcavitysize-1].parents[2] = outcavity[idx].handle;
						}

						/* if this parent has a lesser depth value, update new tet's depth to be the lesser */
						if (outcavity[idx].depth < outcavity[outcavitysize-1].depth)
						{
							outcavity[outcavitysize-1].depth = outcavity[idx].depth;
						}
					}
					else
					{
						/* record this non-wall face for potential addition to F later */
						nonwall[numnonwall++] = i;
					}
				}

				for (i = 0; i < numnonwall; ++i)
				{
					/* this is a newly-uncovered face. there could be more tets behind it, so
					we should add it to F, if the current tet's depth isn't more than the max */
					if (outcavity[outcavitysize-1].depth < CAVDEPTHLIMIT)
					{
						F[Fsize++] = nonwall[i];
					}
					/* we should artificially make this a wall face so the cavity doesn't get deeper */
					else
					{
						/* construct output face */
						cavface2.handle = nonwall[i];
						halfFacePoints(point, halfface, nonwall[i], fpoint);
						cavface2.quality = tetquality(pnew, fpoint[0], fpoint[2], fpoint[1], qualmeasure);
						cavface2.child = NOCAVITYTET;

						/* add it to parent tet */
						if (outcavity[outcavitysize-1].outfacesize < 5)
							outcavity[outcavitysize-1].outfaces[outcavity[outcavitysize-1].outfacesize++] = cavface2;

						W[Wsize++] = nonwall[i];
					}
				}
			}
			else
			{
				/* note that this face has no child, and assign it to its parent tet */
				cavface.child = NOCAVITYTET;
				if (outcavity[tetidx].outfacesize < 5)
					outcavity[tetidx].outfaces[outcavity[tetidx].outfacesize++] = cavface;

				/* add f to W, it borders a blocking tet */
				W[Wsize++] = currf;
			}
			continue;
		}

		/* t is neither a blocking tet nor a cavity tet */
		/* check to see if the three other faces of the tet are facing v */
		bool check = true;
		for (i = 0; i < 3; ++i)
		{
			/* fetch the face points */
			halfFacePoints(point, halfface, otherfaces[i], fpoint);
			/* the order of points should be reversed */
			if (orient(pnew, fpoint[2], fpoint[1], fpoint[0]) != 1)
			{
				check = false;
				break;
			}
		}
		if (check)
		{
			/* yes! we can add this tet to the cavity */
			//C[Csize++] = currtet;

			/* add this tet to the output cavity */
			cavtet.handle = currtet;
			cavtet.outfacesize = 0;
			cavtet.quality = tets[currtet].quality;

			/* it's parent must be the parent above */
			cavtet.parents[0] = outcavity[tetidx].handle;
			/* depth is one deeper than parent */
			cavtet.depth = outcavity[tetidx].depth + 1;
			/* if this is a new deepest, note it */
			if (cavtet.depth > deepest) deepest = cavtet.depth;
			outcavity[outcavitysize++] = cavtet;

			/* note the current face's child in the parent tet */
			cavface.child = currtet;
			if (outcavity[tetidx].outfacesize < 5)
				outcavity[tetidx].outfaces[outcavity[tetidx].outfacesize++] = cavface;

			/* add t's three (outward oriented) faces to F, if the current tet isn't too deep */
			if (cavtet.depth < CAVDEPTHLIMIT)
			{
				F[Fsize++] = otherfaces[0];
				F[Fsize++] = otherfaces[1];
				F[Fsize++] = otherfaces[2];
			}
			else
			{
				/* construct output face */
				cavface2.child = NOCAVITYTET;
				for (i = 0; i < 3; i++)
				{
					cavface2.handle = otherfaces[i];
					halfFacePoints(point, halfface, otherfaces[i], fpoint);
					cavface2.quality = tetquality(pnew, fpoint[0], fpoint[1], fpoint[2], qualmeasure);
					/* add it to parent tet */
					outcavity[outcavitysize-1].outfaces[outcavity[outcavitysize-1].outfacesize++] = cavface2;
				}
				W[Wsize++] = otherfaces[0];
				W[Wsize++] = otherfaces[1];
				W[Wsize++] = otherfaces[2];
			}
		}
		else
		{
			/* this is a blocking tet, add it to B */
			B[Bsize++] = currtet;

			/* note the current face in the parent tet */
			cavface.child = NOCAVITYTET;
			if (outcavity[tetidx].outfacesize < 5)
			{
				outcavity[tetidx].outfaces[outcavity[tetidx].outfacesize++] = cavface;
			}

			/* add the current face to the wall face list */
			W[Wsize++] = currf;
		}
	}

	/* record the maximum depth */
	//cavdeep = deepest;
}

// insert sort
__device__ void sort_cavityedge(CavityEdge *edges, int edgesize)
{
	int i, j;
	CavityEdge tedge;
	for (i = 1; i < edgesize; ++i)
	{
		tedge = edges[i];
		for (j = i; j > 0; --j)
		{
			if (edges[j-1].qual > tedge.qual)
				edges[j] = edges[j-1];
			else
				break;
		}
		edges[j] = tedge;
	}
}


/* recursively label parents and children as cavity tets */
__device__ void cavityLabel(CavityTet *cavity, int cavitysize, int ctetidx_)
{
	int i;
	int parenttetidx;
	int childtetidx;
	int tetstack[MAXSTACKTET];
	int tetstacksize = 0;
	int ctetidx = ctetidx_;

	/* this tet shouldn't yet be labeled */
	if (cavity[ctetidx].label != NOLABEL)
		return;

	tetstack[tetstacksize++] = ctetidx;

	while (tetstacksize > 0)
	{
		ctetidx = tetstack[tetstacksize-1];
		-- tetstacksize;

		if (cavity[ctetidx].label != NOLABEL)
			continue;
		/* label this tet as in the cavity */

		cavity[ctetidx].label = CAVLABEL;

		/* go through all parents in the original graph */
		for (i = 0; i < 3; ++i)
		{
			if (cavity[ctetidx].parents[i] != NOCAVITYTET)
			{
				/* if this parent is unlabeled, label it */
				parenttetidx = findCavityTet(cavity, cavitysize, cavity[ctetidx].parents[i]);
				if (parenttetidx != cavitysize && cavity[parenttetidx].label == NOLABEL)
				{
					tetstack[tetstacksize++] = parenttetidx;
					//cavityLabel(cavity, cavitysize, parenttetidx);
				}
			}
		}

		/* go through all children in H */
		for (i = 0;  i < cavity[ctetidx].outfacesize; ++i)
		{
			/* check if this edge is in H */
			if (cavity[ctetidx].outfaces[i].inH == true)
			{
				/* this can't be an edge leading to t... we should never add those */
				childtetidx = findCavityTet(cavity, cavitysize, cavity[ctetidx].outfaces[i].child);
				if (childtetidx != cavitysize && cavity[childtetidx].label == NOLABEL)
				{
					tetstack[tetstacksize++] = childtetidx;
					//cavityLabel(cavity, cavitysize, childtetidx);
				}
			}
		}
	}
}

/* recursively label parents and children as anti-cavity tets */
__device__ void antiCavityLabel(CavityTet *cavity, int cavitysize, int ctetidx_)
{
	int parenttetidx;
	int childtetidx;
	int parent,edgetochild;
	int i, j;
	int tetstack[MAXSTACKTET];
	int tetstacksize = 0;
	int ctetidx = ctetidx_;

	/* this tet shouldn't yet be labeled */
	if (cavity[ctetidx].label != NOLABEL)
		return;

	tetstack[tetstacksize++] = ctetidx;

	while (tetstacksize > 0)
	{
		ctetidx = tetstack[tetstacksize-1];
		-- tetstacksize;

		if (cavity[ctetidx].label != NOLABEL)
			continue;
		/* label this tet as in the anticavity */
		cavity[ctetidx].label = ANTICAVLABEL;

		/* go through all parents in H */
		for (i = 0; i < 3; ++i)
		{
			parent = cavity[ctetidx].parents[i];

			if (parent != NOCAVITYTET)
			{
				/* is this parent unlabeled ? */
				parenttetidx = findCavityTet(cavity, cavitysize, cavity[ctetidx].parents[i]);
				if (parenttetidx == cavitysize && cavity[parenttetidx].label != NOLABEL)
				{
					continue;
				}

				/* find the edge from this parent down to the child */
				edgetochild = -1;
				for (j = 0; j < cavity[parenttetidx].outfacesize; j++)
				{
					if (cavity[parenttetidx].outfaces[j].child == cavity[ctetidx].handle)
					{
						edgetochild = j;
					}
				}

				/* is this edge in H? */
				if (cavity[parenttetidx].outfaces[edgetochild].inH == true)
				{
					tetstack[tetstacksize++] = parenttetidx;
					//antiCavityLabel(cavity, cavitysize, parenttetidx);
				}
			}
		}

		/* go through all children in original graph G */
		for (i = 0; i < cavity[ctetidx].outfacesize; i++)
		{
			/* if the child is t, it's the end and is already labeled. move on */
			if (cavity[ctetidx].outfaces[i].child == NOCAVITYTET)
			{
				continue;
			}

			childtetidx = findCavityTet(cavity, cavitysize, cavity[ctetidx].outfaces[i].child);
			if (childtetidx != cavitysize && cavity[childtetidx].label == NOLABEL)
			{
				tetstack[tetstacksize++] = childtetidx;
				//antiCavityLabel(cavity, cavitysize, childtetidx);
			}
		}
	}
}


// smooth the inserting vertex with the cavity faces information
__device__ bool smoothInsertVertex(float *points, cu_halfface *halfface, int *cavityfaces, int cavityfacesize, 
								   float p[3], int qualmetric, float &worstcavity)
{
	float newp[3];
	float qualbefore, qualafter;
	float tempqual;
	int neipoint[MAXCAVITYFACES];
	int neipointcd[MAXCAVITYFACES][3];
	int neipointsize = 0;
	int i, j, k;
	float fpoint[3][3];
	int fpidx;
	//double point[3];
	//double **tetp;
	//int tetcnt;
	//VolumeMesh::Nonsmoother nonsmoother;
	qualbefore = qualafter = 1.0;

	// for each face of cavity
	for (i = 0; i < cavityfacesize; i++)
	{
		// for each point of current face
		for (j = 0; j < 3; j++)
		{
			fpidx = halfface[cavityfaces[i]].pointhandle[j];
			// check if it has been recorded
			for (k = 0; k < neipointsize; ++k)
			{
				if (neipoint[k] == fpidx)
					break;
			}
			// if the point has not been recorded
			if (k == neipointsize)
			{
				neipoint[neipointsize++] = fpidx;
				neipointcd[neipointsize-1][0] = points[fpidx*3];
				neipointcd[neipointsize-1][1] = points[fpidx*3+1];
				neipointcd[neipointsize-1][2] = points[fpidx*3+2];
			}
		}

		halfFacePoints(points, halfface, cavityfaces[i], fpoint);
		tempqual = tetquality(p, fpoint[0], fpoint[1], fpoint[2], qualmetric);
		if (qualbefore > tempqual)
			qualbefore = tempqual;
	}

	/* smooth the point */
	newp[0] = newp[1] = newp[2] = 0.0;
	for (i = 0; i < neipointsize; ++i)
	{
		newp[0] += neipointcd[i][0];
		newp[1] += neipointcd[i][1];
		newp[2] += neipointcd[i][2];
	}
	newp[0] /= neipointsize;
	newp[1] /= neipointsize;
	newp[2] /= neipointsize;
	// fetch smoothing information
	//tetcnt = faces.size()/3;
	//tetp = new double *[tetcnt];
	//for (int i = 0; i < tetcnt; i++)
	//{
	//	tetp[i] = new double[9];
	//	for (int j = 0; j < 3; j++)
	//	{
	//		tetp[i][j*3]   = faces[i*3+j][0];
	//		tetp[i][j*3+1] = faces[i*3+j][1];
	//		tetp[i][j*3+2] = faces[i*3+j][2];
	//	}
	//}
	//point[0] = p[0]; point[1] = p[1]; point[2] = p[2];

	// do vertice smoothing
	//nonsmoother.setTmesh(tmesh);
	//nonsmoother.setQualityKind(MINSINE2);
	//nonsmoother.NonsmoothVertice(point, tetp, tetcnt, point);
	/* end of smoothing */

	// calculate local quality after smoothing
	//newp[0] = point[0];newp[1] = point[1];newp[2] = point[2];
	for (i = 0; i < cavityfacesize; i++)
	{
		halfFacePoints(points, halfface, cavityfaces[i], fpoint);
		tempqual = tetquality(p, fpoint[0], fpoint[1], fpoint[2], qualmetric);
		if (qualafter > tempqual)
			qualafter = tempqual;
	}

	if (qualafter > qualbefore)
	{
		p[0] = newp[0];
		p[1] = newp[1];
		p[2] = newp[2];
		worstcavity = qualafter;
		return true;
	}
	worstcavity = qualbefore;
	return false;
}

/* ALGORITHM 3 + CONSIDER DELETED TETS*/
/* given the DAG representing the largest possible cavity,
find the subcavity that has the lexicographically maximum
quality, then dig it out and connect the exposed faces to 
the new vertex, returning the new tets as well as the worst
quality in this cavity */
__device__ void maxCavity(float *points, cu_halfface *halfface, float pnew[3], CavityTet *cavity, int cavitysize,
						  int *erasetetras, int &erasetetrasize, int *outputfaces, int &outputfacesize, 
						  float &worstdelete, float &cavityqual, int qualmeasure)
{
	//bool foundparent = false;                /* did we find face parent? */
	struct CavityEdge edges[MAXCAVITYEDGES];      /* list of all edges in the DAG */
	int numedges = 0;
	//CavityTet t;                   /* the virtual node t, at the end of array */
	int parentlabel, childlabel;             /* the groups that contain parent and child of an edge */
	float depthfactor;
	//float qual;
	//int deepest = 0;
	int parenttetidx, childtetidx;
	CavityEdge tmpedge;
	int i, j;
	float cavitydepthtable[DEPTHTABLESIZE] = {1.0, 1.6, 2.3, 2.9, 3.3, 3.3, 3.3, 3.3, 3.3, 3.3};

	/* initialize t as a ghost tet */
	//t.handle = NOCAVITYTET;

	/* now, proceed through the DAG, recording all edges, and deleting child
	information to remove the edges, producing D' */
	for (i = 0; i < cavitysize; i++)
	{
		/* initialize this tet's label. set s = cavity, others no label */
		if (i == 0)
		{
			cavity[i].label = CAVLABEL;
		}
		else
		{
			cavity[i].label = NOLABEL;
		}

		/* compute how much to exagerrate quality because of depth */
		if (cavity[i].depth < DEPTHTABLESIZE)
		{
			depthfactor = cavitydepthtable[cavity[i].depth];
		}
		else
		{
			depthfactor = cavitydepthtable[DEPTHTABLESIZE - 1];
		}

		/* for each outgoing face */
		for (j = 0; j < cavity[i].outfacesize; ++j)
		{
			tmpedge.label = EDGELABEL;
			tmpedge.parent = cavity[i].handle;
			tmpedge.qual = cavity[i].outfaces[j].quality * depthfactor;
			/* save which outgoing face this child was */
			tmpedge.childnum = j;

			if (cavity[i].outfaces[j].child == NOCAVITYTET)
			{
				/* if it has no child tet, make up a virtual edge to t */
				tmpedge.child = GHOSTTET;
			}
			else
			{
				/* set edge child to be actual child */
				tmpedge.child = cavity[i].outfaces[j].child;
			}
			edges[numedges++] = tmpedge;

			/* initialize H by setting no edges to be in it */
			cavity[i].outfaces[j].inH = false;
		}
	}

	/* now, sort the edges in order of ascending quality */
	sort_cavityedge(edges, numedges);

	//cuPrintf("\nsorted edges %d :\n", numedges);
	//for (i = 0; i < numedges; i++)
	//{
	//	cuPrintf("edge %d : qual(%f) label(%d) parent(%d) child(%d) childnum(%d)\n", 
	//		     i, edges[i].qual, edges[i].label, edges[i].parent, edges[i].child, edges[i].childnum);
	//}

	/* go through each edge, adding it to D' if it doesn't
	connect s to t */
	for (i = 0; i < numedges; i++)
	{
		/* find parent cavity */
		parenttetidx = findCavityTet(cavity, cavitysize, edges[i].parent);

		/* check parent's label */
		parentlabel = cavity[parenttetidx].label;
		/* check child's label */
		childtetidx = findCavityTet(cavity, cavitysize, edges[i].child);
		if (edges[i].child == GHOSTTET)
			/* this child is labeled automatically as anti-cavity */
			childlabel = ANTICAVLABEL;
		else
			childlabel = cavity[childtetidx].label;

		/* if the parent is in the cavity */
		if (parentlabel == CAVLABEL)
		{
			/* and the child is in the anti-cavity */
			if (childlabel == ANTICAVLABEL)
			{
				/* record output face from parent to child, record the half face of child */
				outputfaces[outputfacesize++] = cavity[parenttetidx].outfaces[edges[i].childnum].handle;
			}
			/* otherwise, if the child isn't labeled */
			else
			{
				if (childlabel == NOLABEL)
				{
					cavityLabel(cavity, cavitysize, childtetidx);
				}
			}
		}
		/* parent isn't labeled cavity. */
		else
		{
			/* is the parent wholly unlabeled ? */
			if (parentlabel == NOLABEL)
			{
				/* is the child labeled anti-cavity ? */
				if (childlabel == ANTICAVLABEL)
				{
					antiCavityLabel(cavity, cavitysize, parenttetidx);
				}
				else
					/* neither the parent nor the child is labeled */
				{
					/* add the edge from parent to child to H */
					cavity[parenttetidx].outfaces[edges[i].childnum].inH = true;
				}
			}
		}
	}

	/* keep track of what the deepest tet in the final cavity was */
	//deepest = 0;

	/* delete all tets labeled as cavity */
	worstdelete = HUGEFLOAT;
	for (i = 0; i < cavitysize; i++)
	{
		if (cavity[i].label == CAVLABEL)
		{
			/* is this the worst quality tet we're deleting? */
			if (cavity[i].quality < worstdelete)
			{
				worstdelete = cavity[i].quality;
			}

			/* is this the deepest tet we've encountered? */
			//if (cavity[i].depth > deepest)
			//{
			//	deepest = cavity[i].depth;
			//}
			erasetetras[erasetetrasize++]= cavity[i].handle;
		}
	}

	// smooth insert point
	smoothInsertVertex(points, halfface, outputfaces, outputfacesize, pnew, qualmeasure, cavityqual);
}



__global__ void vertex_Insertion_Explore(float *points, int pointcnt, cu_InsertTet *tets, int tetcnt, cu_face *face, int facecnt, 
										 cu_halfface *halfface, int halffacecnt, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	float quality;
	cu_InsertTet currtet;
	float tetpoint[4][3];
	int i;

	while(tid < tetcnt)
	{
		//if (tid != 1653)
		//{
		//	tid += offset;
		//	continue;
		//}
		//if (tid != 1439 && tid != 2263 && tid != 1412)
		//{
		//	tid += offset;
		//	continue;
		//}

		//cuPrintf("\n\ncurrent tid : %d\n", tid);

		currtet = tets[tid];

		// calculate current tet's quality
		int pidx;
		for (i = 0; i < 4; i++)
		{
			pidx = currtet.v[i];
			tetpoint[i][0] = points[3*pidx];
			tetpoint[i][1] = points[3*pidx+1];
			tetpoint[i][2] = points[3*pidx+2];
		}
		quality = tetquality(tetpoint, qualmeasure);
		tets[tid].quality = quality;
		currtet.quality = quality;

		//cuPrintf("\ntet vertex : %d %d %d %d\n", currtet.v[0], currtet.v[1], currtet.v[2], currtet.v[3]);
		//cuPrintf("tet quality : %f\n", currtet.quality);

		// select tetra with bad quality to do vertex insertion
		if (quality > VERTEX_INSERTION_QUALITY_THRESHOLD)
		{
			tid += offset;
			continue;
		}

		//cuPrintf("\nThis tet is a target tet\n");

		int outcavitysize = 0;
		CavityTet outcavity[MAXCAVITYTETS];
		float worstdelete = HUGEFLOAT;
		float cavityqual = HUGEFLOAT;

		/* build the cavity dag */
		float barycenter[3];
		for (i = 0; i < 3; i++)
		{
			barycenter[i] = (tetpoint[0][i] + tetpoint[1][i] + tetpoint[2][i] + tetpoint[3][i])/4.0;
		}

		//cuPrintf("\nCenter point : %f %f %f\n", barycenter[0], barycenter[1], barycenter[2]);

		buildcavitydag(points, tets, face, halfface, barycenter, tid, currtet, tetpoint, qualmeasure, outcavity, outcavitysize);

		//cuPrintf("\nout cavity tet %d :\n", outcavitysize);
		//for (i = 0; i < outcavitysize; i++)
		//{
		//	cuPrintf("%d ", outcavity[i].handle);
		//}
		//cuPrintf("\n");


		/* build the cavity of maximum lexicographic quality */
		int outputfaces[MAXCAVITYFACES];
		int outputfacesize = 0;
		int erasetetras[MAXCAVITYTETS];  /* tetras which are going to be erased */
		int erasetetrasize = 0;
		maxCavity(points, halfface, barycenter, outcavity, outcavitysize, erasetetras, erasetetrasize, 
			      outputfaces, outputfacesize, worstdelete, cavityqual, qualmeasure);

		//cuPrintf("\nerase tet %d :\n", erasetetrasize);
		//for (i = 0; i < erasetetrasize; i++)
		//{
		//	cuPrintf("%d ", erasetetras[i]);
		//}
		//cuPrintf("\n");


		//cuPrintf("\ncavity face %d :\n", outputfacesize);
		//for (i = 0; i < outputfacesize; i++)
		//{
		//	cuPrintf("%d ", outputfaces[i]);
		//}
		//cuPrintf("\n");

		//cuPrintf("\n new point : %f %f %f\n", barycenter[0], barycenter[1], barycenter[2]);

		//cuPrintf("\nquality before: %f     quality after: %f\n", worstdelete, cavityqual);

		/* did we succeed? */
		if (cavityqual > (worstdelete + MINIMPROVEMENT))
		{
			tets[tid].deletetetcnt = erasetetrasize;
			for (i = 0; i < erasetetrasize; i++)
			{
				tets[tid].deletetet[i] = erasetetras[i];
			}
			tets[tid].cavityfacecnt = outputfacesize;
			for (i = 0; i < outputfacesize; i++)
			{
				tets[tid].cavityface[i] = outputfaces[i];
			}
			tets[tid].insertpoint[0] = barycenter[0];
			tets[tid].insertpoint[1] = barycenter[1];
			tets[tid].insertpoint[2] = barycenter[2];
			tets[tid].val = worstdelete;
		}
		tid += offset;
	}
}

__global__ void vertex_insertion_tetquality(float *points, cu_InsertTet *tets, int tetcnt, int qualmeasure)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	float tetpoint[4][3];
	int pidx;

	while(tid < tetcnt)
	{
		// if the tet is invalid
		if (tets[tid].v[0] == -1)
		{
			tid += offset;
			continue;
		}

		// get tetra points
		for (int j = 0; j < 4; j++)
		{
			pidx = tets[tid].v[j];
			tetpoint[j][0] = points[3*pidx];
			tetpoint[j][1] = points[3*pidx+1];
			tetpoint[j][2] = points[3*pidx+2];
		}
		// calculate tetra quality
		tets[tid].quality = tetquality(tetpoint, qualmeasure);

		tid += offset;
	}
}

void inserTet_selection(cu_InsertTet *tets, int tetcnt, int *selecttet, int &selecttetcnt, int &maxincreasetet)
{
	int stetcnt = 0;
	int* stet = new int[tetcnt];
	int i, j;
	int temp;
	maxincreasetet = 0;

	/* pick out the tets by vertex insertion result
	and sort them*/
	for (i = 0; i < tetcnt; i++)
	{
		if (tets[i].deletetetcnt == 0)
			continue;

		// insert the tet
		for (j = stetcnt-1; j > -1 ; j--)
		{
			if (tets[stet[j]].val > tets[i].val)
				stet[j+1] = stet[j];
			else 
				break;
		}
		stet[j+1] = i;
		++ stetcnt;
	}

	/* if there are no any tet meeting the requirement */
	if (!stetcnt)
	{
		selecttetcnt = 0;
		selecttet = NULL;
		return;
	}

	/* select a new tet set in which the cavities of any two of them are not overlapped */
	bool isok;
	bool *tetflag;
	tetflag = new bool[tetcnt];
	memset(tetflag, 1, tetcnt*sizeof(bool));

	selecttet = new int[stetcnt];
	// push the first tet
	selecttet[0] = stet[0];
	selecttetcnt = 1;

	// get increase tet num
	temp = tets[stet[0]].cavityfacecnt - tets[stet[0]].deletetetcnt;
	if (temp > maxincreasetet)
		maxincreasetet = temp;

	// set flags of cavity tets
	for (i = 0; i < tets[stet[0]].deletetetcnt; i++)
	{
		tetflag[tets[stet[0]].deletetet[i]] = 0;
	}

	/* if the cavity tets of current tetra are available, 
	then add the tet into the array*/
	for (i = 1; i < stetcnt; i++)
	{
		isok = true;
		for (j = 0; j < tets[stet[i]].deletetetcnt; j++)
		{
			if (tetflag[tets[stet[i]].deletetet[j]] == 0)
			{
				isok = false;
				break;
			}
		}

		if (isok)
		{
			selecttet[selecttetcnt++] = stet[i];
			for (j = 0; j < tets[stet[i]].deletetetcnt; j++)
			{
				tetflag[tets[stet[i]].deletetet[j]] = 0;
			}

			// get increase tet num
			temp = tets[stet[i]].cavityfacecnt - tets[stet[i]].deletetetcnt;
			if (temp > maxincreasetet)
				maxincreasetet = temp;
		}
	}
}

// δ����face��halfface�������޸�
__global__ void vertex_insertion(float *points, int pointcnt, cu_InsertTet *tets, cu_halfface *halfface, 
								 int *selecttet, int selecttetcnt, int *increasetets, int maxincreasetet)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	cu_InsertTet currtet;
	int pidx;
	int tetidx;
	int tetnum;
	int i;

	while(tid < selecttetcnt)
	{
		currtet = tets[selecttet[tid]];

		// add new vertex
		pidx = pointcnt + tid;
		points[pidx*3]   = currtet.insertpoint[0];
		points[pidx*3+1] = currtet.insertpoint[1];
		points[pidx*3+2] = currtet.insertpoint[2];

		// add new tet
		if (currtet.deletetetcnt < currtet.cavityfacecnt)
			tetnum = currtet.deletetetcnt;
		else
			tetnum = currtet.cavityfacecnt;

		for (i = 0; i < tetnum; i++)
		{
			tetidx = currtet.deletetet[i];
			tets[tetidx].v[0] = pidx;
			tets[tetidx].v[1] = halfface[currtet.cavityface[i]].pointhandle[0];
			tets[tetidx].v[2] = halfface[currtet.cavityface[i]].pointhandle[1];
			tets[tetidx].v[3] = halfface[currtet.cavityface[i]].pointhandle[2];
		}

		if (currtet.deletetetcnt < currtet.cavityfacecnt)
		{
			for (i = currtet.deletetetcnt; i < currtet.cavityfacecnt; i++)
			{
				tetidx = (maxincreasetet*tid + (i-currtet.deletetetcnt))*4;
				increasetets[tetidx]   = pidx;
				increasetets[tetidx+1] = halfface[currtet.cavityface[i]].pointhandle[0];
				increasetets[tetidx+2] = halfface[currtet.cavityface[i]].pointhandle[1];
				increasetets[tetidx+3] = halfface[currtet.cavityface[i]].pointhandle[2];
			}
		}
		else
		{
			for (i = currtet.cavityfacecnt; i < currtet.deletetetcnt; i++)
			{
				tetidx = currtet.deletetet[i];
				tets[tetidx].v[0] = -1;
				tets[tetidx].v[1] = -1;
				tets[tetidx].v[2] = -1;
				tets[tetidx].v[3] = -1;
			}
		}

		tid += offset;
	}
}

extern "C" void cuda_vertexInsertion(float *&points, int pointcnt, int *meshtets, int tetcnt, int *face, int facecnt, 
									 int *halfface, int halffacecnt, int qualmeasure, float& qualbefore_, float& qualafter_, 
									 int &succ, float &time)
{
	//FILE *outfile = NULL;
	//outfile = fopen("F:\\kernel_data_output.txt", "w");

	//fprintf(outfile, "In cuda vertexinsertion\n");

	// create face and halfface structure
	//int *meshtets;
	struct cu_face *cface;
	struct cu_halfface *chalfface;
	struct cu_InsertTet *ctet;
	float qualbefore;
	float qualafter;
	int tetcapacity;
	int facecapacity;
	int halffacecapacity;
	int pointcapacity;

	tetcapacity = int(1.1*tetcnt);
	facecapacity = int(1.1*facecnt);
	halffacecapacity = int(1.1*halffacecnt);
	pointcapacity = int(1.1*pointcnt);

	//meshtets = new int[4*tetcapacity];
	ctet = new struct cu_InsertTet[tetcapacity];
	cface = new struct cu_face[facecapacity];
	chalfface = new struct cu_halfface[halffacecapacity];

	//memcpy(meshtets, meshtets_, 4*tetcnt*sizeof(int));
	for (int i = 0; i < tetcnt; i ++)
	{
		ctet[i].v[0] = meshtets[4*i];
		ctet[i].v[1] = meshtets[4*i+1];
		ctet[i].v[2] = meshtets[4*i+2];
		ctet[i].v[3] = meshtets[4*i+3];
		ctet[i].deletetetcnt = 0;
		memset(ctet[i].deletetet, -1, MAXCAVITYTETS*sizeof(int));
		ctet[i].cavityfacecnt = 0;
		memset(ctet[i].cavityface, -1, MAXCAVITYFACES*sizeof(int));
		ctet[i].quality = 1.0;
	}

	// calculate quality before flipping
	//cuda_tetquality(points, pointcnt, meshtets, tetcnt, qualmeasure, qualbefore);

	// CUDA Parallel
	// �����豸�洢�ռ�
	//int loop = 1;
	float *dev_points;
	struct cu_InsertTet *dev_tets;
	struct cu_face *dev_face/*, *dev_tempface*/;
	struct cu_halfface *dev_halfface/*, *dev_temphalfface*/;
	int *dev_selecttet;
	int *dev_increasetets;

	int *selecttet = NULL;
	int selecttetcnt = 0;
	int *increasetet = NULL;
	int maxincreasetet = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int i = 0; i < facecnt; i++)
	{
		cface[i].hf[0] = face[2*i];
		cface[i].hf[1] = face[2*i+1];
	}

	for (int i = 0; i < halffacecnt; i++)
	{
		chalfface[i].pointhandle[0] = halfface[4*i];
		chalfface[i].pointhandle[1] = halfface[4*i+1];
		chalfface[i].pointhandle[2] = halfface[4*i+2];
		chalfface[i].face = halfface[4*i+3];
	}


	// �����������µ������塢������棬�����Ԥ����һ���ֿռ�
	hipMalloc((void**)&dev_points, 3*pointcapacity*sizeof(float));
	hipMalloc((void**)&dev_tets, tetcapacity*sizeof(cu_InsertTet));
	hipMalloc((void**)&dev_face, facecapacity*sizeof(cu_face));
	hipMalloc((void**)&dev_halfface, halffacecapacity*sizeof(cu_halfface));

	hipMemcpy(dev_points, points, 3*pointcnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_tets, ctet, tetcnt*sizeof(cu_InsertTet), hipMemcpyHostToDevice);
	hipMemcpy(dev_face, cface, facecnt*sizeof(cu_face), hipMemcpyHostToDevice);
	hipMemcpy(dev_halfface, chalfface, halffacecnt*sizeof(cu_halfface), hipMemcpyHostToDevice);

	// vertex insertion explore
	// ���ú˺���

	//InitGPUSet();
	//cuPrintInit();

	// step 1 : vertex insertion explore
	//int blocks = imin(tetcnt, BlockPerGrid);
	int blocks = min (tetcnt, BlockPerGrid);
	int threads = min((tetcnt+blocks-1)/blocks, ThreadPerBlock);
	vertex_insertion_tetquality<<<blocks, threads>>>(dev_points, dev_tets, tetcnt, qualmeasure);
	vertex_Insertion_Explore<<<blocks, threads>>>(dev_points, pointcnt, dev_tets, tetcnt, dev_face, facecnt, dev_halfface, halffacecnt, qualmeasure);

	//cudaPrintfDisplay(outfile, false);
	//cudaPrintfEnd(); 
	//fclose(outfile);

	// step 2 : tetra selection
	// get updated tetra data
	hipMemcpy(ctet, dev_tets, tetcnt*sizeof(cu_InsertTet), hipMemcpyDeviceToHost);
	inserTet_selection(ctet, tetcnt, selecttet, selecttetcnt, maxincreasetet);

	// step 3 : do parallel vertex insertion
	if (selecttetcnt != 0)
	{
		/** check if point set has enough space
		    if not enlarge the point set*/
		if ((selecttetcnt+pointcnt) > pointcapacity)
		{
			while((selecttetcnt+pointcnt) > pointcapacity)
				pointcapacity = int(1.1*pointcapacity);

			float *dev_temp_points;
			hipMalloc((void**)&dev_temp_points, 3*pointcnt*sizeof(float));
			hipMemcpy(dev_temp_points, dev_points, 3*pointcnt*sizeof(float), hipMemcpyDeviceToDevice);
			hipFree(dev_points);
			hipMalloc((void**)&dev_points, 3*pointcapacity*sizeof(float));
			hipMemcpy(dev_points, dev_temp_points, 3*pointcnt*sizeof(float), hipMemcpyDeviceToDevice);
			hipFree(dev_temp_points);
		}

		// do vertex insertion
		hipMalloc((void**)&dev_selecttet, selecttetcnt*sizeof(int));
		hipMemcpy(dev_selecttet, selecttet, selecttetcnt*sizeof(int), hipMemcpyDeviceToDevice);

		// apply space for the increasing tets
		
		hipMalloc((void**)&dev_increasetets, maxincreasetet*selecttetcnt*4*sizeof(int));
		hipMemset(dev_increasetets, -1, maxincreasetet*selecttetcnt*4*sizeof(int));

		blocks = min (selecttetcnt, BlockPerGrid);
		threads = min((selecttetcnt+blocks-1)/blocks, ThreadPerBlock);
		vertex_insertion<<<blocks, threads>>>(dev_points, pointcnt, dev_tets, dev_halfface, 
			                                  dev_selecttet, selecttetcnt, dev_increasetets, maxincreasetet);

		pointcnt += selecttetcnt;
		delete [] selecttet;
	}


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elaspsedTime;
	hipEventElapsedTime(&elaspsedTime, start, stop);

	time = elaspsedTime;

	// calculate quality after flipping
	qualafter = 1.0;
	if (selecttetcnt)
	{
		delete [] points;
		points = new float[pointcnt*3];
		hipMemcpy(points, dev_points, 3*pointcnt*sizeof(float), hipMemcpyDeviceToHost);
		
		increasetet = new int [maxincreasetet*selecttetcnt*4];
		hipMemcpy(increasetet, dev_increasetets, maxincreasetet*selecttetcnt*4*sizeof(int), hipMemcpyDeviceToHost);
		cuda_tetquality(points, pointcnt, increasetet, maxincreasetet*selecttetcnt, qualmeasure, qualafter);
		delete [] increasetet;
	}
	vertex_insertion_tetquality<<<blocks, (tetcnt+blocks-1)/blocks>>>(dev_points, dev_tets, tetcnt, qualmeasure);
	hipMemcpy(ctet, dev_tets, tetcnt*sizeof(cu_InsertTet), hipMemcpyDeviceToHost);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(dev_points);
	hipFree(dev_tets);
	hipFree(dev_halfface);
	hipFree(dev_face);

	for (int i = 0; i < tetcnt; i++)
	{
		if (ctet[i].quality < qualafter)
			qualafter = ctet[i].quality;
	}
	qualbefore = 1.0;
	qualbefore_ = qualbefore;
	qualafter_ = qualafter;
	succ = selecttetcnt;

	delete [] ctet;
	delete [] cface;
	delete [] chalfface;
	//delete [] increasetet;
}
/******************** End of Parallel Vertex Insertion ************************/